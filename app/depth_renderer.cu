#include "core/soa.cuh"
#include "core/camera_model.cuh"
#include "renderer/depth.cuh"
#include <ext/lodepng/lodepng.h>

__constant__ DeviceCamera dev_cam;

int main() {
    InitProfiler();

    // right, down, back, left, up
    int num_triangle = 10, num_spheres = 3, num_prims = num_triangle + num_spheres;
    int spp       = 2;
    std::vector<Vec3> v1s = {{1, 1, 1}, {1, 1, 1}, {-1, 1, -1}, {-1, 1, -1}, {-1, 1, 1}, {-1, 1, 1}, {-1, -1, 1}, {-1, 1, 1}, {-1,-1, 1}, {-1, -1, 1}, {0.5, 0, -0.7}, {-0.4,0.4, -0.5}, {-0.5, -0.5, -0.7}};
    std::vector<Vec3> v2s = {{1,-1,-1}, {1, -1,1}, {1, 1,  -1}, {1, -1, -1}, {1, 1,  1}, {1, 1, -1}, {-1, 1,  1}, {-1, 1,-1}, { 1,-1, 1}, {1,  1,  1}, {0.3, 0, 0}, {0.5, 0, 0}, {0.3, 0, 0}};
    std::vector<Vec3> v3s = {{1, 1,-1}, {1,-1,-1}, {1, -1, -1}, {-1, -1,-1}, {1, 1, -1}, {-1,1, -1}, {-1, -1,-1}, {-1,-1,-1}, { 1, 1, 1}, {-1, 1,  1}, {0, 0, 0}, {0, 0, 0}, {0, 0, 0}};
    Vec3 normal_default = {0, 1, 0};
    Vec2 uv_default     = {0.5, 0.5};

    // scene setup
    SoA3<Vec3> vert_data(v1s.size()), norm_data(v1s.size());
    SoA3<Vec2> uvs_data(v1s.size());
    vert_data.from_vectors(v1s, v2s, v3s);
    norm_data.fill(normal_default);
    uvs_data.fill(uv_default);

    // camera setup
    Vec3 from(0, -3, 0), to(0, 0, 0);
    int width = 1024, height = 1024;
    float fov = 55;
    DeviceCamera camera(from, to, fov, width, height);
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(dev_cam), &camera, sizeof(DeviceCamera)));

    // shape setup
    std::vector<Shape> shapes(num_prims);
    for (int i = 0; i < num_triangle; i++)
        shapes[i] = TriangleShape(i >> 1);
    for (int i = num_triangle; i < num_prims; i++)
        shapes[i] = SphereShape(i >> 1);
    
    DepthTracer dtracer(shapes, vert_data, norm_data, uvs_data, width, height);
    auto bytes_buffer = dtracer.render(spp);

    std::string file_name = "depth-render.png";

    if (unsigned error = lodepng::encode(file_name, bytes_buffer, width, height); error) {
        std::cerr << "lodepng::encoder error " << error << ": " << lodepng_error_text(error)
                  << std::endl;
        throw std::runtime_error("lodepng::encode() fail");
    }

    printf("image saved to `%s`\n", file_name.c_str());

    vert_data.destroy();
    norm_data.destroy();
    uvs_data.destroy();

    // ReportThreadStats();    
    // PrintStats(stdout);
    ReportProfilerResults(stdout);

    // ClearStats();
    ClearProfiler();
    CleanupProfiler();
    return 0;
}