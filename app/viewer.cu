#include <sstream>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <ext/imgui/imgui.h>
#include <ext/imgui/backends/imgui_impl_glfw.h>
#include <ext/imgui/backends/imgui_impl_opengl3.h>

#include "core/xyz.cuh"
#include "core/scene.cuh"
#include "core/serialize.h"
#include "core/imgui_utils.cuh"

#include "renderer/bvh_cost.cuh"
#include "renderer/light_tracer.cuh"
#include "renderer/wf_path_tracer.cuh"

CPT_GPU_CONST Emitter* c_emitter[9];
CPT_GPU_CONST BSDF*    c_material[48];

std::string get_current_time() {
    // Get the current time as a time_point
    auto now = std::chrono::system_clock::now();

    // Convert to time_t to extract time components
    std::time_t now_time = std::chrono::system_clock::to_time_t(now);
    std::tm* local_time = std::localtime(&now_time);

    // Use stringstream to format the output
    std::stringstream oss;
    oss << std::put_time(local_time, "%Y-%m-%d-%H-%M-%S");

    return oss.str();
}

int main(int argc, char** argv) {
    CUDA_CHECK_RETURN(hipFree(nullptr));       // initialize CUDA
    omp_set_num_threads(4);

    std::cerr << "[MAIN] Path tracing IMGUI viewer.\n";
    if (argc < 2) {
        std::cerr << "Input file not specified. Usage: ./pt <path to xml>\n";
        exit(1);
    }
    std::string xml_path = argv[1];

    std::cout << "[SCENE] Loading scenes from '" << xml_path << "'\n";
    Scene scene(xml_path);
    
    ColorSpaceXYZ xyz_host;
    xyz_host.init();
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_material), scene.bsdfs, scene.num_bsdfs * sizeof(BSDF*)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_emitter), scene.emitters, (scene.num_emitters + 1) * sizeof(Emitter*)));
#ifdef TRIANGLE_ONLY
    printf("[ATTENTION] Note that TRIANGLE_ONLY macro is defined. Please make sure there is no sphere primitive in the scene.\n");
#endif
    std::unique_ptr<TracerBase> renderer = nullptr;
    std::cout << "[RENDERER] Path tracer loaded: ";
    switch (scene.rdr_type) {
        case RendererType::MegaKernelPT: {
            renderer = std::make_unique<PathTracer>(scene); 
            std::cout << "\tMegakernel Path Tracing.\n";
            break;
        }
        case RendererType::WavefrontPT: {
            renderer = std::make_unique<WavefrontPathTracer>(scene);
            std::cout << "\tWavefront Path Tracing..\n";
            break;
        }
        case RendererType::MegeKernelLT: {
            renderer = std::make_unique<LightTracer>(scene, scene.config.spec_constraint,
                        scene.config.bidirectional, scene.config.caustic_scaling); 
            if (scene.config.bidirectional)
                std::cout << "\tNaive Bidirectional Megakernel Light Tracing.\n";
            else
                std::cout << "\tMegakernel Light Tracing.\n";
            break;
        } 
        case RendererType::VoxelSDFPT: {
            std::cerr << "\tVoxelSDFPT is not implemented yet. Stay tuned. Rendering exits.\n";
            return 0;
        }
        case RendererType::DepthTracing: {
            renderer = std::make_unique<DepthTracer>(scene);
            std::cerr << "\tDepth Tracing\n";
            break;
        }
        case RendererType::BVHCostViz: {
            renderer = std::make_unique<BVHCostVisualizer>(scene);
            std::cerr << "\tBVH Cost Visualizer\n";
            break;
        }
        default: {
            throw std::runtime_error("Unsupported renderer type.");
        }
    }

    scene.free_resources();

    auto window = gui::create_window(scene.config.width, scene.config.height);
    renderer->graphics_resc_init(gui::init_texture_and_pbo);
    renderer->update_camera(scene.cam);
    gui::GUIParams params;
    Serializer::push<int>(params.serialized_data, 1);
    params.gamma_corr   = scene.config.gamma_correction;
    bool exit_main_loop = false;

    ImGuiIO& io = ImGui::GetIO();

    while (!glfwWindowShouldClose(window.get())) {
        glfwPollEvents();

        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        gui::show_render_statistics(
            renderer->get_num_sample() + 1,
            params.show_fps
        );
        params.reset();
        params.camera_update = gui::keyboard_camera_update(*scene.cam, params.trans_speed, params.capture, exit_main_loop);
        if (exit_main_loop) {
            break;
        }
        gui::render_settings_interface(
            *scene.cam, scene.emitter_props, scene.bsdf_infos, scene.config.md, params, scene.rdr_type
        );
        if (!io.WantCaptureMouse) {        // no sub window (setting window or main menu) is focused
            params.camera_update |= gui::mouse_camera_update(*scene.cam, params.rot_sensitivity);
        }
        if (params.scene_update) {
            scene.update_emitters();
            CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_emitter), scene.emitters, (scene.num_emitters + 1) * sizeof(Emitter*)));
        }
        if (params.material_update) {
            scene.update_materials();
            CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_material), scene.bsdfs, scene.num_bsdfs * sizeof(BSDF*)));
        }
        if (params.camera_update) {
            renderer->update_camera(scene.cam);
        }
        if (params.serialized_update) {
            renderer->param_setter(params.serialized_data);
        }
        if (params.buffer_flush_update())
            renderer->reset_out_buffer();
        renderer->render_online(scene.config.md, params.gamma_corr);
        
        if (params.capture) {
            auto fbuffer = renderer->get_image_buffer(params.gamma_corr);
            std::string format = params.output_png ? "png" : "jpg";
            std::string file_name = "render-" + get_current_time() + "." + format;
            
            if (!save_image(file_name, fbuffer, scene.config.width, scene.config.height, format, params.compress_q)) {
                std::cerr << "stb::save_image() failed to output image" << std::endl;
                throw std::runtime_error("stb::save_image() fail");
            } else {
                std::cout << "[Viewer] Image file saved to '" << file_name << "'\n";
            }
        }
        gui::update_texture(
            renderer->get_pbo_id(),
            renderer->get_texture_id(),
            scene.config.width,
            scene.config.height
        );
        gui::window_render(
            renderer->get_texture_id(),
            scene.config.width,
            scene.config.height
        );

        // swap the buffer
        glfwSwapBuffers(window.get());
    }
    scene.print();
    gui::clean_up(
        window.get(),
        renderer->get_pbo_id(),
        renderer->get_texture_id()
    );
    xyz_host.destroy();

    return 0;
}