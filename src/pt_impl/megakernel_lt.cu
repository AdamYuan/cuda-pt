#include "hip/hip_runtime.h"
/**
 * Megakernel Light Tracing (Implementation)
 * Note that though LT and PT are both declared in `megakernel_pt.cuh`
 * We separate LT and PT implementation, for the sake of clarity
 * Also, I only intend to implement the megakernel version
 * Since LT is not so tile-based, WF ideas are less intuitive
 * just for me
 * 
 * @date: 9.28.2024
 * @author: Qianyue He
*/
#include "renderer/megakernel_pt.cuh"

static constexpr int RR_BOUNCE = 2;
static constexpr float RR_THRESHOLD = 0.1;

/**
 * @brief this version does not employ object-level culling
 * we use shared memory to accelerate rendering instead, for object-level culling
 * shared memory might not be easy to use, since the memory granularity will be
 * too difficult to control
 * 
 * @param objects   object encapsulation
 * @param verts     vertices
 * @param norms     normal vectors, ArrayType: (p1, 3D) -> (p2, 3D) -> (p3, 3D)
 * @param uvs       uv coordinates, ArrayType: (p1, 2D) -> (p2, 2D) -> (p3, 2D)
 * @param camera    GPU camera model (constant memory)
 * @param image     GPU image buffer
 * @param max_depth maximum allowed bounce
*/
template <bool render_once>
CPT_KERNEL void render_lt_kernel(
    const DeviceCamera& dev_cam, 
    const PrecomputedArray verts,
    const NormalArray norms, 
    const ConstBuffer<PackedHalf2> uvs,
    ConstObjPtr objects,
    ConstIndexPtr emitter_prims,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t nodes,
    ConstF4Ptr cached_nodes,
    DeviceImage image,
    const MaxDepthParams md_params,
    float* __restrict__ output_buffer,
    int num_emitter,
    int seed_offset,
    int node_num,
    int accum_cnt,
    int cache_num,
    int specular_constraints,
    float caustic_scale,
    bool gamma_corr
) {
    int px = threadIdx.x + blockIdx.x * blockDim.x, py = threadIdx.y + blockIdx.y * blockDim.y;
    int constraint_cnt = 0;

    Sampler sampler(px + py * image.w(), seed_offset);
    // step 1: generate ray (sample ray from one emitter and sample from that emitter)
    Ray ray;
    Vec4 throughput;
    {
        uint32_t emitter_id = (sampler.discrete1D() % uint32_t(num_emitter)) + 1;
        Emitter* emitter = c_emitter[emitter_id];
        float emitter_sample_pdf = 1.f / float(num_emitter), le_pdf = 1.f;

        Vec2 extras = sampler.next2D();
        emitter_id = objects[emitter->get_obj_ref()].sample_emitter_primitive(sampler.discrete1D(), le_pdf);
        emitter_id = emitter_prims[emitter_id];
        throughput = emitter->sample_le(ray.o, ray.d, le_pdf, sampler.next2D(), verts, norms, uvs, emitter_id, extras.x(), extras.y());
        throughput *= 1.f / (emitter_sample_pdf * le_pdf);
    }

    // step 2: bouncing around the scene until the max depth is reached
    int min_index = -1, diff_b = 0, spec_b = 0, trans_b = 0;

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    extern __shared__ uint4 s_cached[];
    if (tid < cache_num) {      // no more than 256 nodes will be cached
        s_cached[tid] = cached_nodes[tid];
        int offset_tid = tid + blockDim.x * blockDim.y;
        if (offset_tid < cache_num)
            s_cached[offset_tid] = cached_nodes[offset_tid];
    }
    __syncthreads();

    for (int b = 0; b < md_params.max_depth; b++) {
        float prim_u = 0, prim_v = 0, min_dist = MAX_DIST;

        int min_object_info = INVALID_OBJ;
        min_index = -1;
        // ============= step 1: ray intersection =================
        min_dist = ray_intersect_bvh(
            ray, bvh_leaves, nodes, s_cached, 
            verts, min_index, min_object_info, 
            prim_u, prim_v, node_num, cache_num, min_dist
        );

        bool is_triangle = true;
        int object_id = extract_object_info(min_object_info, is_triangle);

        // ============= step 2: local shading for indirect bounces ================
        if (min_index >= 0) {
            auto it = Primitive::get_interaction(verts, norms, uvs, ray.advance(min_dist), prim_u, prim_v, min_index, is_triangle);

            // ============= step 3: next event estimation ================
            // (1) randomly pick one emitter
            int material_id = 0, dummy = -1;
            objects[object_id].unpack(material_id, dummy);

            // deterministically connect to the camera
            Ray shadow_ray(ray.advance(min_dist), Vec3(0, 0, 1));
            shadow_ray.d = dev_cam.t - shadow_ray.o;
            float emit_len_mis = shadow_ray.d.length();
            shadow_ray.d *= __frcp_rn(emit_len_mis);              // normalized direction

            // (3) Light tracing NEE scene intersection test (possible warp divergence, but... nevermind)
            int pixel_x = -2, pixel_y = -2;
            if (constraint_cnt > specular_constraints &&
                dev_cam.get_splat_pixel(shadow_ray.d, pixel_x, pixel_y) && 
                occlusion_test_bvh(shadow_ray, bvh_leaves, nodes, s_cached, 
                        verts, node_num, cache_num, emit_len_mis - EPSILON)
            ) {
                Vec4 direct_splat = throughput * c_material[material_id]->eval(it, shadow_ray.d, ray.d, material_id, false, false) * \
                    (float(emit_len_mis > EPSILON) * __frcp_rn(emit_len_mis < EPSILON ? 1.f : emit_len_mis));
                auto& to_write = image(pixel_x, pixel_y);
                atomicAdd(&to_write.x(), direct_splat.x() * caustic_scale);
                atomicAdd(&to_write.y(), direct_splat.y() * caustic_scale);
                atomicAdd(&to_write.z(), direct_splat.z() * caustic_scale);
                atomicAdd(&to_write.w(), 1.f);
            }

            // step 4: sample a new ray direction, bounce the 
            ray.o = std::move(shadow_ray.o);
            ScatterStateFlag sampled_lobe = ScatterStateFlag::BSDF_NONE;
            ray.d = c_material[material_id]->sample_dir(ray.d, it, throughput, emit_len_mis, sampler, sampled_lobe, material_id, false);
            constraint_cnt += c_material[material_id]->require_lobe(ScatterStateFlag::BSDF_SPECULAR);

            // step 5: russian roulette
            diff_b  += (ScatterStateFlag::BSDF_DIFFUSE  & sampled_lobe) > 0;
            spec_b  += (ScatterStateFlag::BSDF_SPECULAR & sampled_lobe) > 0;
            trans_b += (ScatterStateFlag::BSDF_TRANSMIT & sampled_lobe) > 0;
            if (diff_b  >= md_params.max_diffuse  || 
                spec_b  >= md_params.max_specular || 
                trans_b >= md_params.max_tranmit
            ) break;
            float max_value = throughput.max_elem_3d();
            if (b >= RR_BOUNCE && max_value < RR_THRESHOLD) {
                if (sampler.next1D() > max_value || max_value < THP_EPS) break;
                throughput *= 1. / max_value;
            }
            // using BVH enables breaking, since there is no within-loop synchronization
        }
    }
    __syncthreads();
    if constexpr (render_once) {
        // image will be the output buffer, there will be double buffering
        Vec4 radiance = image(px, py);
        radiance *= 1.f / float(accum_cnt);
        radiance = gamma_corr ? radiance.gamma_corr() : radiance;
        FLOAT4(output_buffer[(px + py * image.w()) << 2]) = float4(radiance); 
    }
}

template CPT_KERNEL void render_lt_kernel<true>(
    const DeviceCamera& dev_cam, 
    const PrecomputedArray verts,
    const NormalArray norms, 
    const ConstBuffer<PackedHalf2> uvs,
    ConstObjPtr objects,
    ConstIndexPtr emitter_prims,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t nodes,
    ConstF4Ptr cached_nodes,
    DeviceImage image,
    const MaxDepthParams md_params,
    float* __restrict__ output_buffer,
    int num_emitter,
    int seed_offset,
    int node_num,
    int accum_cnt,
    int cache_num,
    int specular_constraints,
    float caustic_scale,
    bool gamma_corr
);

template CPT_KERNEL void render_lt_kernel<false>(
    const DeviceCamera& dev_cam, 
    const PrecomputedArray verts,
    const NormalArray norms, 
    const ConstBuffer<PackedHalf2> uvs,
    ConstObjPtr objects,
    ConstIndexPtr emitter_prims,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t nodes,
    ConstF4Ptr cached_nodes,
    DeviceImage image,
    const MaxDepthParams md_params,
    float* __restrict__ output_buffer,
    int num_emitter,
    int seed_offset,
    int node_num,
    int accum_cnt,
    int cache_num,
    int specular_constraints,
    float caustic_scale,
    bool gamma_corr
);