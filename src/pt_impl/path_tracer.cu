#include "hip/hip_runtime.h"
/**
 * @file path_tracer.cu
 * @author Qianyue He
 * @brief Megakernel Path Tracer implementation
 * @date 2024.10.10
 * @copyright Copyright (c) 2024
 */

#include "renderer/path_tracer.cuh"

static constexpr int SHFL_THREAD_X = 4;     // blockDim.x: 1 << SHFL_THREAD_X, by default, SHFL_THREAD_X is 4: 16 threads
static constexpr int SHFL_THREAD_Y = 3;     // blockDim.y: 1 << SHFL_THREAD_Y, by default, SHFL_THREAD_Y is 4: 16 threads

PathTracer::PathTracer(
    const Scene& scene
): TracerBase(scene), 
    num_objs(scene.objects.size()), num_nodes(-1), num_emitter(scene.num_emitters), 
    cuda_texture_id(0), pbo_id(0), output_buffer(nullptr), accum_cnt(0)
{
#ifdef RENDERER_USE_BVH
    if (scene.bvh_available()) {
        size_t num_bvh  = scene.obj_idxs.size();
        num_nodes = scene.node_fronts.size();
        num_cache = scene.cache_fronts.size();
        CUDA_CHECK_RETURN(hipMalloc(&_obj_idxs,  num_bvh * sizeof(int)));
        CUDA_CHECK_RETURN(hipMalloc(&_node_fronts, num_nodes * sizeof(float4)));
        CUDA_CHECK_RETURN(hipMalloc(&_node_backs,  num_nodes * sizeof(float4)));
        CUDA_CHECK_RETURN(hipMalloc(&_cached_nodes, 2 * num_cache * sizeof(float4)));
        // note that BVH leaf node only stores the primitive to object mapping
        PathTracer::createTexture1D<int>(scene.obj_idxs.data(), num_bvh, _obj_idxs, bvh_leaves);
        PathTracer::createTexture1D<float4>(scene.node_fronts.data(), num_nodes, _node_fronts, node_fronts);
        PathTracer::createTexture1D<float4>(scene.node_backs.data(),  num_nodes, _node_backs,  node_backs);
        CUDA_CHECK_RETURN(hipMemcpy(_cached_nodes, scene.cache_fronts.data(), sizeof(float4) * num_cache, hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpy(&_cached_nodes[num_cache], scene.cache_backs.data(), sizeof(float4) * num_cache, hipMemcpyHostToDevice));
    } else {
        throw std::runtime_error("BVH not available in scene. Abort.");
    }
#endif  // RENDERER_USE_BVH
    size_t emitter_prim_size = sizeof(int) * scene.emitter_prims.size();
    CUDA_CHECK_RETURN(hipMallocManaged(&obj_info, num_objs * sizeof(ObjInfo)));
    CUDA_CHECK_RETURN(hipMalloc(&camera, sizeof(DeviceCamera)));
    CUDA_CHECK_RETURN(hipMalloc(&emitter_prims, emitter_prim_size));
    CUDA_CHECK_RETURN(hipMemcpy(camera, &scene.cam, sizeof(DeviceCamera), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(emitter_prims, scene.emitter_prims.data(), emitter_prim_size, hipMemcpyHostToDevice));
    for (int i = 0; i < num_objs; i++)
        obj_info[i] = scene.objects[i];
#ifdef TRIANGLE_ONLY
    printf("[ATTENTION] Note that TRIANGLE_ONLY macro is defined. Please make sure there is no sphere primitive in the scene.\n");
#endif
}

PathTracer::~PathTracer() {
    CUDA_CHECK_RETURN(hipFree(obj_info));
    CUDA_CHECK_RETURN(hipFree(camera));
    CUDA_CHECK_RETURN(hipFree(emitter_prims));
#ifdef RENDERER_USE_BVH
    CUDA_CHECK_RETURN(hipDestroyTextureObject(bvh_leaves));
    CUDA_CHECK_RETURN(hipDestroyTextureObject(node_fronts));
    CUDA_CHECK_RETURN(hipDestroyTextureObject(node_backs));
    CUDA_CHECK_RETURN(hipFree(_obj_idxs));
    CUDA_CHECK_RETURN(hipFree(_node_fronts));
    CUDA_CHECK_RETURN(hipFree(_node_backs));
    CUDA_CHECK_RETURN(hipFree(_cached_nodes));
#endif  // RENDERER_USE_BVH
    printf("[Renderer] Path Tracer Object destroyed.\n");
}

CPT_CPU std::vector<uint8_t> PathTracer::render(
    int num_iter,
    int max_depth,
    bool gamma_correction
) {
    printf("Rendering starts.\n");
    TicToc _timer("render_pt_kernel()", num_iter);
    size_t cached_size = std::max(2 * num_cache * sizeof(float4), sizeof(float4));
    for (int i = 0; i < num_iter; i++) {
        // for more sophisticated renderer (like path tracer), shared_memory should be used
        render_pt_kernel<false><<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y), dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
            *camera, verts, norms, uvs, obj_info, aabbs, emitter_prims,
            bvh_leaves, node_fronts, node_backs, _cached_nodes,
            image, output_buffer, num_prims, num_objs, num_emitter, 
            i * SEED_SCALER, max_depth, num_nodes, accum_cnt, num_cache
        ); 
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
        printProgress(i, num_iter);
    }
    printf("\n");
    return image.export_cpu(1.f / num_iter, gamma_correction);
}

CPT_CPU void PathTracer::render_online(
    int max_depth
) {
    CUDA_CHECK_RETURN(hipGraphicsMapResources(1, &pbo_resc, 0));
    size_t _num_bytes = 0, cached_size = std::max(2 * num_cache * sizeof(float4), sizeof(float4));
    CUDA_CHECK_RETURN(hipGraphicsResourceGetMappedPointer((void**)&output_buffer, &_num_bytes, pbo_resc));

    accum_cnt ++;
    render_pt_kernel<true><<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y), dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
        *camera, verts, norms, uvs, obj_info, aabbs, emitter_prims, 
        bvh_leaves, node_fronts, node_backs, _cached_nodes,
        image, output_buffer, num_prims, num_objs, num_emitter, 
        accum_cnt * SEED_SCALER, max_depth, num_nodes, accum_cnt, num_cache
    ); 
    CUDA_CHECK_RETURN(hipGraphicsUnmapResources(1, &pbo_resc, 0));
}