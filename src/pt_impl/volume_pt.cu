#include "hip/hip_runtime.h"
// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Megakernel Volumetric Path Tracer implementation
 * @date 2025.2.9
 */
#include "renderer/megakernel_vpt.cuh"
#include "renderer/volume_pt.cuh"

static constexpr int SEED_SCALER =
    11467; // 11451 is not a prime, while 11467 is
static constexpr int SHFL_THREAD_X =
    5; // blockDim.x: 1 << SHFL_THREAD_X, by default, SHFL_THREAD_X is 4: 16
       // threads
static constexpr int SHFL_THREAD_Y =
    2; // blockDim.y: 1 << SHFL_THREAD_Y, by default, SHFL_THREAD_Y is 4: 16
       // threads

VolumePathTracer::VolumePathTracer(const Scene &scene)
    : PathTracer(scene, false), cam_vol_id(scene.cam_vol_id) {
    media = scene.media;
}

VolumePathTracer::~VolumePathTracer() {
    printf("[Renderer] Volume Path Tracer Object destroyed.\n");
}

CPT_CPU std::vector<uint8_t> VolumePathTracer::render(const MaxDepthParams &md,
                                                      int num_iter,
                                                      bool gamma_correction) {
    printf("Rendering starts.\n");
    TicToc _timer("render_pt_kernel()", num_iter);
    size_t cached_size = std::max(num_cache * sizeof(uint4), sizeof(uint4));
    for (int i = 0; i < num_iter; i++) {
        // for more sophisticated renderer (like path tracer), shared_memory
        // should be used
        render_vpt_kernel<false>
            <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
               dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
                *camera, verts, norms, uvs, media, obj_info, emitter_prims,
                bvh_leaves, nodes, _cached_nodes, image, md, output_buffer,
                nullptr, num_emitter, i * SEED_SCALER + seed_offset, cam_vol_id,
                num_nodes, accum_cnt, num_cache, envmap_id);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
        printProgress(i, num_iter);
    }
    printf("\n");
    return image.export_cpu(1.f / num_iter, gamma_correction);
}

CPT_CPU void VolumePathTracer::render_online(const MaxDepthParams &md,
                                             bool gamma_corr) {
    CUDA_CHECK_RETURN(hipGraphicsMapResources(1, &pbo_resc, 0));
    size_t _num_bytes = 0,
           cached_size = std::max(num_cache * sizeof(uint4), sizeof(uint4));
    // if we have an illegal memory access here: check whether you have a valid
    // emitter in the xml scene description file. it might be possible that
    // having no valid emitter triggers an illegal memory access
    CUDA_CHECK_RETURN(hipGraphicsResourceGetMappedPointer(
        (void **)&output_buffer, &_num_bytes, pbo_resc));
    accum_cnt++;

    render_vpt_kernel<true>
        <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
           dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
            *camera, verts, norms, uvs, media, obj_info, emitter_prims,
            bvh_leaves, nodes, _cached_nodes, image, md, output_buffer, nullptr,
            num_emitter, accum_cnt * SEED_SCALER + seed_offset, cam_vol_id,
            num_nodes, accum_cnt, num_cache, envmap_id, gamma_corr);
    CUDA_CHECK_RETURN(hipGraphicsUnmapResources(1, &pbo_resc, 0));
}

CPT_CPU const float *VolumePathTracer::render_raw(const MaxDepthParams &md,
                                                  bool gamma_corr) {
    size_t cached_size = std::max(num_cache * sizeof(uint4), sizeof(uint4));
    accum_cnt++;
    render_vpt_kernel<true>
        <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
           dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
            *camera, verts, norms, uvs, media, obj_info, emitter_prims,
            bvh_leaves, nodes, _cached_nodes, image, md, output_buffer,
            var_buffer, num_emitter, accum_cnt * SEED_SCALER + seed_offset,
            cam_vol_id, num_nodes, accum_cnt, num_cache, envmap_id, gamma_corr);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    return output_buffer;
}
