#include "hip/hip_runtime.h"
/**
 * Wavefront Path Tracing (Implementation)
 * @date: 9.15.2024
 * @author: Qianyue He
*/
#include "renderer/wavefront_pt.cuh"

namespace {
    using PayLoadBuffer      = PayLoadBufferSoA* const __restrict__;
    using ConstPayLoadBuffer = const PayLoadBuffer;
}

static constexpr int SEED_SCALER = 11451;       //-4!
static constexpr int RR_BOUNCE = 2;
static constexpr float RR_THRESHOLD = 0.1;

/**
 * @brief ray generation kernel 
 * note that, all the kernels are called per stream, each stream can have multiple blocks (since it is a kernel call)
 * let's say, for example, a 4 * 4 block for one kernel call. These 16 blocks should be responsible for 
 * one image patch, offseted by the stream_offset.
 * @note we first consider images that have width and height to be the multiple of 128
 * to avoid having to consider the border problem
 * @note we pass payloads in by value
*/ 
CPT_KERNEL void raygen_primary_hit_shader(
    const DeviceCamera& dev_cam,
    PayLoadBufferSoA payloads,
    const PrecomputedArray verts,
    const ArrayType<Vec3> norms, 
    const ConstBuffer<PackedHalf2> uvs,
    ConstObjPtr objects,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t nodes,
    ConstF4Ptr cached_nodes,
    const IndexBuffer idx_buffer,
    int stream_offset, int num_prims,
    int x_patch, int y_patch, int iter,
    int stream_id, int width, 
    int node_num, int cache_num
) {
    // stream and patch related offset
    const int sx = x_patch * PATCH_X, sy = y_patch * PATCH_Y, buffer_xoffset = stream_id * PATCH_X;
    // within kernel thread position
    const int px = threadIdx.x + blockIdx.x * blockDim.x, py = threadIdx.y + blockIdx.y * blockDim.y;
    // linear idx_buffer position
    const int block_index = py * blockDim.x * gridDim.x + px;

    Sampler sg = Sampler(px + sx + (py + sy) * width, iter * SEED_SCALER);
    Ray ray = dev_cam.generate_ray(px + sx, py + sy, sg.next2D());

    Interaction it;                          // To local register

    int min_index = -1, min_object_id = 0;   // round up
    ray.hit_t = MAX_DIST;

    #ifdef FUSED_MISS_SHADER
    ray.set_active(false);
    #endif   // FUSED_MISS_SHADER
    float prim_u = 0, prim_v = 0;

    payloads.thp(px + buffer_xoffset, py) = Vec4(1, 1, 1, 1);
    idx_buffer[block_index + stream_id * TOTAL_RAY] = (py << 16) + px + buffer_xoffset;    
        // cache near root level BVH nodes for faster traversal
    extern __shared__ float4 s_cached[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < 2 * cache_num) {      // no more than 128 nodes will be cached
        s_cached[tid] = cached_nodes[tid];
    }
    __syncthreads();
    ray.hit_t = ray_intersect_bvh(ray, bvh_leaves, nodes, 
                    s_cached, verts, min_index, min_object_id, 
                    prim_u, prim_v, node_num, cache_num, MAX_DIST);

    // ============= step 2: local shading for indirect bounces ================
    payloads.L(px + buffer_xoffset, py)   = Vec4(0, 0, 0, 1);
    payloads.set_sampler(px + buffer_xoffset, py, sg);
    if (min_index >= 0) {
        // if the ray hits nothing, or the path throughput is 0, then the ray will be inactive
        // inactive rays will only be processed in the miss_shader
        ray.set_hit();
        ray.set_hit_index(min_index);
#ifdef FUSED_MISS_SHADER
        ray.set_active(true);
#endif   // FUSED_MISS_SHADER
        it = Primitive::get_interaction(verts, norms, uvs, ray.advance(ray.hit_t), prim_u, prim_v, min_index, min_object_id >= 0);
    }

    // compress two int (to int16) to a uint32_t 
    // note that we can not use int here, since int shifting might retain the sign
    // it is implementation dependent
    // note that we only have stream_number * payloadbuffers
    // so row indices won't be offset by sy, col indices should only be offseted by stream_offset
    payloads.set_ray(px + buffer_xoffset, py, ray);
    payloads.interaction(px + buffer_xoffset, py) = it;
     
    // px has already encoded stream_offset (stream_id * PATCH_X)
}

/**
 * @brief find ray intersection for next hit pos
 * We first start with small pool size (4096), which can comprise at most 16 blocks
 * The ray pool is stream-compacted (with thrust::parition to remove the finished)
 * Note that we need an index buffer, since the Ray and Sampler are coupled
 * and we need the index to port the 
*/ 
CPT_KERNEL void closesthit_shader(
    PayLoadBufferSoA payloads,
    const PrecomputedArray verts,
    const ArrayType<Vec3> norms, 
    const ConstBuffer<PackedHalf2> uvs,
    ConstObjPtr objects,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t nodes,
    ConstF4Ptr cached_nodes,
    const IndexBuffer idx_buffer,
    int stream_offset,
    int num_prims,
    int num_valid,
    int node_num,
    int cache_num
) {
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px

    uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
    py >>= 16;
    Ray        ray = payloads.get_ray(px, py);
    Interaction it = payloads.interaction(px, py);                           // To local register
    ray.reset();
    
    float prim_u = 0, prim_v = 0;
    int min_index = -1, min_object_id = 0;   // round up
    ray.hit_t = MAX_DIST;

    // cache near root level BVH nodes for faster traversal
    extern __shared__ float4 s_cached[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < 2 * cache_num) {      // no more than 128 nodes will be cached
        s_cached[tid] = cached_nodes[tid];
    }
    __syncthreads();
    ray.hit_t = ray_intersect_bvh(ray, bvh_leaves, nodes, 
                    s_cached, verts, min_index, min_object_id, 
                    prim_u, prim_v, node_num, cache_num, MAX_DIST);

    // ============= step 2: local shading for indirect bounces ================
    if (block_index < num_valid && min_index >= 0) {
        // if the ray hits nothing, or the path throughput is 0, then the ray will be inactive
        // inactive rays will only be processed in the miss_shader
        ray.set_hit();
        ray.set_hit_index(min_index);
#ifdef FUSED_MISS_SHADER
        ray.set_active(true);
#endif   // FUSED_MISS_SHADER
        it = Primitive::get_interaction(verts, norms, uvs, ray.advance(ray.hit_t), prim_u, prim_v, min_index, min_object_id >= 0);
    }

    payloads.set_ray(px, py, ray);
    payloads.interaction(px, py) = it;
}

/***
 * For non-delta hit (shading point), direct component should be evaluated:
 * we sample a light source then start ray intersection test
*/
CPT_KERNEL void nee_shader(
    PayLoadBufferSoA payloads,
    const PrecomputedArray verts,
    const ArrayType<Vec3> norms, 
    const ConstBuffer<PackedHalf2> uvs,
    ConstObjPtr objects,
    ConstIndexPtr emitter_prims,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t nodes,
    ConstF4Ptr cached_nodes,
    const IndexBuffer idx_buffer,
    int stream_offset,
    int num_prims,
    int num_objects,
    int num_emitter,
    int num_valid,
    int node_num,
    int cache_num
) {
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px
    // cache near root level BVH nodes for faster traversal
    extern __shared__ float4 s_cached[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < 2 * cache_num) {      // no more than 128 nodes will be cached
        s_cached[tid] = cached_nodes[tid];
    }
    __syncthreads();
    
    if (block_index < num_valid) {
        uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
        py >>= 16;
        Vec4 thp = payloads.thp(px, py);
        Ray ray  = payloads.get_ray(px, py);
        Sampler sg = payloads.get_sampler(px, py);
        const Interaction it = payloads.interaction(px, py);

        int object_id = tex1Dfetch<int>(bvh_leaves, ray.hit_id());
        object_id = object_id >= 0 ? object_id : -object_id - 1;        // sphere object ID is -id - 1
        int material_id = 0, emitter_id = -1;
            objects[object_id].unpack(material_id, emitter_id);

        float direct_pdf = 1;

        Emitter* emitter = sample_emitter(sg, direct_pdf, num_emitter, emitter_id);
        emitter_id = objects[emitter->get_obj_ref()].sample_emitter_primitive(sg.discrete1D(), direct_pdf);
        emitter_id = emitter_prims[emitter_id];               // extra mapping, introduced after BVH primitive reordering
        Ray shadow_ray(ray.advance(ray.hit_t), Vec3(0, 0, 0));
        // use ray.o to avoid creating another shadow_int variable
        Vec4 direct_comp(0, 0, 0, 1);
        shadow_ray.d = emitter->sample(shadow_ray.o, it.shading_norm, direct_comp, direct_pdf, sg.next2D(), verts, norms, uvs, emitter_id) - shadow_ray.o;

        float emit_len_mis = shadow_ray.d.length();
        shadow_ray.d *= __frcp_rn(emit_len_mis);              // normalized direct
        // (3) NEE scene intersection test (possible warp divergence, but... nevermind)
        if (emitter != c_emitter[0] && 
            occlusion_test_bvh(shadow_ray, bvh_leaves, nodes, 
                    s_cached, verts, node_num, cache_num, emit_len_mis - EPSILON)
        ) {
            // MIS for BSDF / light sampling, to achieve better rendering
            // 1 / (direct + ...) is mis_weight direct_pdf / (direct_pdf + material_pdf), divided by direct_pdf
            emit_len_mis = direct_pdf + c_material[material_id]->pdf(it, shadow_ray.d, ray.d, material_id) * emitter->non_delta();
            payloads.L(px, py) += thp * direct_comp * c_material[material_id]->eval(it, shadow_ray.d, ray.d, material_id) * \
                (float(emit_len_mis > EPSILON) * __frcp_rn(emit_len_mis < EPSILON ? 1.f : emit_len_mis));
            // numerical guard, in case emit_len_mis is 0
        }

        payloads.set_sampler(px, py, sg);
    }
}

/**
 * BSDF sampling & direct shading shader
*/
CPT_KERNEL void bsdf_local_shader(
    PayLoadBufferSoA payloads,
    const hipTextureObject_t bvh_leaves,
    ConstObjPtr objects,
    const IndexBuffer idx_buffer,
    int stream_offset,
    int num_prims, 
    int num_valid,
    bool secondary_bounce
) {
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px

    if (block_index < num_valid) {
        uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
        py >>= 16;

        Vec4 thp = payloads.thp(px, py);
        Ray ray  = payloads.get_ray(px, py);
        Sampler sg = payloads.get_sampler(px, py);
        Interaction it = payloads.interaction(px, py);

        // this is incorrect, since AABB should be reordered, too
        int object_id = tex1Dfetch<int>(bvh_leaves, ray.hit_id());
        object_id = object_id >= 0 ? object_id : -object_id - 1;        // sphere object ID is -id - 1
        int material_id = 0, emitter_id = -1;
            objects[object_id].unpack(material_id, emitter_id);
        
        bool hit_emitter = emitter_id > 0;

        // emitter MIS
        float pdf = payloads.pdf(px, py), emission_weight = pdf / (pdf + 
                objects[object_id].solid_angle_pdf(c_textures.eval_normal(it, material_id), ray.d, ray.hit_t) * 
                hit_emitter * secondary_bounce * ray.non_delta());
        // (2) check if the ray hits an emitter
        Vec4 direct_comp = thp *\
                    c_emitter[emitter_id]->eval_le(&ray.d, &it);
        payloads.L(px, py) += direct_comp * emission_weight;
        
        ray.o = ray.advance(ray.hit_t);
        BSDFFlag sampled_lobe = BSDFFlag::BSDF_NONE;                            
        ray.d = c_material[material_id]->sample_dir(
            ray.d, it, thp, pdf, sg, sampled_lobe, material_id
        );
        ray.set_delta((sampled_lobe & BSDFFlag::BSDF_SPECULAR) > 0);

        payloads.set_sampler(px, py, sg);
        payloads.thp(px, py) = thp;
        payloads.set_ray(px, py, ray);
        payloads.interaction(px, py) = it;
        payloads.pdf(px, py) = pdf;
    }
}

/**
 * Purpose of the miss shader: if ray hits nothing in closesthit shader
 * the we will set the hit status (flag) to be false
 * in this shader, we find the rays marked as no-hit, and check the
 * availability of environment map (currently not supported)
 * after processing the env-map lighting, we mark the ray as inactive
 * before stream compaction. Then stream compaction will 'remove' all these
 * rays (and the threads)
 * 
 * MISS_SHADER is the only place where you mark a ray as inactive
*/
CPT_KERNEL void miss_shader(
    PayLoadBufferSoA payloads,
    const IndexBuffer idx_buffer,
    const int bounce,
    int stream_offset,
    int num_valid,
    int envmap_id
) {
    // Nothing here, currently, if we decide not to support env lighting
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px
    if (block_index < num_valid) {
        uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
        py >>= 16;
        Vec4 thp        = payloads.thp(px, py), old_thp = thp;
        Sampler sampler = payloads.get_sampler(px, py);
        // using BVH enables the usage of RR, since there is no within-loop synchronization
        float max_value = thp.max_elem_3d();

        if (bounce >= RR_BOUNCE && max_value < RR_THRESHOLD) {
            max_value = (sampler.next1D() > max_value || max_value < THP_EPS) ? 0 : max_value;
            thp *= max_value == 0 ? 0 : (1.f / max_value);
        }

        Vec3 ray_d;
        bool is_active = true;
        payloads.get_ray_d(px, py, ray_d, is_active);
        if (is_active && !payloads.is_hit(px, py)) {
            payloads.L(px, py) += old_thp * c_emitter[envmap_id]->eval_le(&ray_d);
            payloads.set_active(px, py, false);
        } else if (max_value <= 1e-5f) {
            payloads.set_active(px, py, false);
        }
        payloads.thp(px, py) = thp;
        payloads.set_sampler(px, py, sampler);
    }
}

template <bool render_once>
CPT_KERNEL void radiance_splat(
    PayLoadBufferSoA payloads, DeviceImage image, 
    int stream_id, int x_patch, int y_patch, 
    int accum_cnt, float* output_buffer, bool gamma_corr
) {
    // Nothing here, currently, if we decide not to support env lighting
    const int px = threadIdx.x + blockIdx.x * blockDim.x, py = threadIdx.y + blockIdx.y * blockDim.y;
    Vec4 L = payloads.L(px + stream_id * PATCH_X, py);         // To local register
    L = L.numeric_err() ? Vec4(0, 0, 0, 1) : L;

    if constexpr (render_once) {
        // image will be the output buffer, there will be double buffering
        int img_x = px + x_patch * PATCH_X, img_y = py + y_patch * PATCH_Y;
        auto local_v = image(img_x, img_y) + L;
        image(img_x, img_y) = local_v;
        local_v *= 1.f / float(accum_cnt);
        local_v = gamma_corr ? local_v.gamma_corr() : local_v;
        FLOAT4(output_buffer[(img_x + img_y * image.w()) << 2]) = float4(local_v); 
    } else {
        image(px + x_patch * PATCH_X, py + y_patch * PATCH_Y) += L;
    }
}

template CPT_KERNEL void radiance_splat<true>(
    PayLoadBufferSoA payloads, DeviceImage image, 
    int stream_id, int x_patch, int y_patch,
    int accum_cnt, float* output_buffer, bool gamma_corr
);

template CPT_KERNEL void radiance_splat<false>(
    PayLoadBufferSoA payloads, DeviceImage image, 
    int stream_id, int x_patch, int y_patch, 
    int accum_cnt, float* output_bufferr, bool gamma_corr
);