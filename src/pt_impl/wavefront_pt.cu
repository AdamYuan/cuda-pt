#include "hip/hip_runtime.h"
/**
 * Wavefront Path Tracing (Implementation)
 * @date: 9.15.2024
 * @author: Qianyue He
*/
#include "renderer/base_pt.cuh"
#include "renderer/wavefront_pt.cuh"

namespace {
    using PayLoadBuffer      = PayLoadBufferSoA* const __restrict__;
    using ConstPayLoadBuffer = const PayLoadBuffer;
}

/**
 * @brief ray generation kernel 
 * note that, all the kernels are called per stream, each stream can have multiple blocks (since it is a kernel call)
 * let's say, for example, a 4 * 4 block for one kernel call. These 16 blocks should be responsible for 
 * one image patch, offseted by the stream_offset.
 * @note we first consider images that have width and height to be the multiple of 128
 * to avoid having to consider the border problem
 * @note we pass payloads in by value
*/ 
CPT_KERNEL void raygen_primary_hit_shader(
    const DeviceCamera& dev_cam,
    const PrecomputedArray& verts,
    PayLoadBufferSoA payloads,
    ConstObjPtr objects,
    ConstAABBPtr aabbs,
    ConstNormPtr norms, 
    ConstUVPtr uvs,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t node_fronts,
    const hipTextureObject_t node_backs,
    ConstF4Ptr cached_nodes,
    const IndexBuffer idx_buffer,
    int stream_offset, int num_prims,
    int x_patch, int y_patch, int iter,
    int stream_id, int width, 
    int node_num, int cache_num
) {
    // stream and patch related offset
    const int sx = x_patch * PATCH_X, sy = y_patch * PATCH_Y, buffer_xoffset = stream_id * PATCH_X;
    // within kernel thread position
    const int px = threadIdx.x + blockIdx.x * blockDim.x, py = threadIdx.y + blockIdx.y * blockDim.y;
    // linear idx_buffer position
    const int block_index = py * blockDim.x * gridDim.x + px;

    Sampler sg = Sampler(px + sx + (py + sy) * width, iter * SEED_SCALER);
    Ray ray = dev_cam.generate_ray(px + sx, py + sy, sg.next2D());

    PDFInteraction it;            // To local register

    int min_index = -1, min_object_id = 0;   // round up
    ray.hit_t = MAX_DIST;

    #ifdef FUSED_MISS_SHADER
    ray.set_active(false);
    #endif   // FUSED_MISS_SHADER
    float prim_u = 0, prim_v = 0;

    payloads.thp(px + buffer_xoffset, py) = Vec4(1, 1, 1, 1);
    idx_buffer[block_index + stream_id * TOTAL_RAY] = (py << 16) + px + buffer_xoffset;    
#ifdef RENDERER_USE_BVH 
        // cache near root level BVH nodes for faster traversal
    extern __shared__ float4 s_cached[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < 2 * cache_num) {      // no more than 128 nodes will be cached
        s_cached[tid] = cached_nodes[tid];
    }
    __syncthreads();
    ray.hit_t = ray_intersect_bvh(ray, bvh_leaves, node_fronts, 
                    node_backs, s_cached, verts, min_index, 
                    min_object_id, prim_u, prim_v, node_num, cache_num, MAX_DIST);
#else   // RENDERER_USE_BVH
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    __shared__ Vec4 s_verts[TRI_IDX(BASE_ADDR)];                // vertex info
    __shared__ AABBWrapper s_aabbs[BASE_ADDR];                  // aabb

    PrecomputedArray s_verts_arr(reinterpret_cast<Vec4*>(&s_verts[0]), BASE_ADDR);
    int num_copy = (num_prims + BASE_ADDR - 1) / BASE_ADDR;

    // ============= step 1: ray intersection =================
    #pragma unroll
    for (int cp_base = 0; cp_base < num_copy; ++cp_base) {
        // memory copy to shared memory
        int cur_idx = (cp_base << BASE_SHFL) + tid, remain_prims = min(num_prims - (cp_base << BASE_SHFL), BASE_ADDR);
        cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
        if (tid < BASE_ADDR && cur_idx < num_prims) {        // copy from gmem to smem
            // we should pad this, for every 3 Vec3, we pad one more vec3, then copy can be made
            // without branch (memcpy_async): TODO, this is the bottle neck. L2 Global excessive here
            // since our step is Vec3, this will lead to uncoalesced access
            // shared memory is enough. Though padding is not easy to implement
            cuda::memcpy_async(&s_verts[TRI_IDX(tid)], &verts.data[TRI_IDX(cur_idx)], sizeof(Vec4) * 3, pipe);
            // This memory op is not fully coalesced, since AABB container is not a complete SOA
            s_aabbs[tid].aabb.copy_from(aabbs[cur_idx]);
        }
        pipe.producer_commit();
        pipe.consumer_wait();
        __syncthreads();
        // this might not be a good solution
        ray.hit_t = ray_intersect(s_verts_arr, ray, s_aabbs, remain_prims, 
                cp_base << BASE_SHFL, min_index, min_object_id, prim_u, prim_v, ray.hit_t);
        __syncthreads();
    }
#endif  // RENDERER_USE_BVH
    // ============= step 2: local shading for indirect bounces ================
    payloads.L(px + buffer_xoffset, py)   = Vec4(0, 0, 0, 1);
    payloads.set_sampler(px + buffer_xoffset, py, sg);
    if (min_index >= 0) {
        // if the ray hits nothing, or the path throughput is 0, then the ray will be inactive
        // inactive rays will only be processed in the miss_shader
        ray.set_hit();
        ray.set_hit_index(min_index);
#ifdef FUSED_MISS_SHADER
        ray.set_active(true);
#endif   // FUSED_MISS_SHADER
        it.it() = Primitive::get_interaction(verts, *norms, *uvs, ray.advance(ray.hit_t), prim_u, prim_v, min_index, min_object_id >= 0);
    }

    // compress two int (to int16) to a uint32_t 
    // note that we can not use int here, since int shifting might retain the sign
    // it is implementation dependent
    // note that we only have stream_number * payloadbuffers
    // so row indices won't be offset by sy, col indices should only be offseted by stream_offset
    payloads.set_ray(px + buffer_xoffset, py, ray);
    payloads.set_interaction(px + buffer_xoffset, py, it);
     
    // px has already encoded stream_offset (stream_id * PATCH_X)
}

/**
 * @brief find ray intersection for next hit pos
 * We first start with small pool size (4096), which can comprise at most 16 blocks
 * The ray pool is stream-compacted (with thrust::parition to remove the finished)
 * Note that we need an index buffer, since the Ray and Sampler are coupled
 * and we need the index to port the 
*/ 
CPT_KERNEL void closesthit_shader(
    const PrecomputedArray& verts,
    PayLoadBufferSoA payloads,
    ConstObjPtr objects,
    ConstAABBPtr aabbs,
    ConstNormPtr norms, 
    ConstUVPtr uvs,
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t node_fronts,
    const hipTextureObject_t node_backs,
    ConstF4Ptr cached_nodes,
    const IndexBuffer idx_buffer,
    int stream_offset,
    int num_prims,
    int num_valid,
    int node_num,
    int cache_num
) {
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px

    uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
    py >>= 16;
    Ray           ray = payloads.get_ray(px, py);
    PDFInteraction it = payloads.get_interaction(px, py);            // To local register
    ray.reset();
    
    float prim_u = 0, prim_v = 0;
    int min_index = -1, min_object_id = 0;   // round up
    ray.hit_t = MAX_DIST;

#ifdef RENDERER_USE_BVH 
    // cache near root level BVH nodes for faster traversal
    extern __shared__ float4 s_cached[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < 2 * cache_num) {      // no more than 128 nodes will be cached
        s_cached[tid] = cached_nodes[tid];
    }
    __syncthreads();
    ray.hit_t = ray_intersect_bvh(ray, bvh_leaves, node_fronts, 
                    node_backs, s_cached, verts, min_index, 
                    min_object_id, prim_u, prim_v, node_num, cache_num, MAX_DIST);
#else   // RENDERER_USE_BVH
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    __shared__ Vec4 s_verts[TRI_IDX(BASE_ADDR)];                // vertex info
    __shared__ AABBWrapper s_aabbs[BASE_ADDR];                  // aabb

    PrecomputedArray s_verts_arr(reinterpret_cast<Vec4*>(&s_verts[0]), BASE_ADDR);
    int num_copy = (num_prims + BASE_ADDR - 1) / BASE_ADDR;

    // ============= step 1: ray intersection =================
    #pragma unroll
    for (int cp_base = 0; cp_base < num_copy; ++cp_base) {
        // memory copy to shared memory
        int cur_idx = (cp_base << BASE_SHFL) + tid, remain_prims = min(num_prims - (cp_base << BASE_SHFL), BASE_ADDR);
        cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
        if (tid < BASE_ADDR && cur_idx < num_prims) {        // copy from gmem to smem
    
            // we should pad this, for every 3 Vec3, we pad one more vec3, then copy can be made
            // without branch (memcpy_async): TODO, this is the bottle neck. L2 Global excessive here
            // since our step is Vec3, this will lead to uncoalesced access
            // shared memory is enough. Though padding is not easy to implement
            cuda::memcpy_async(&s_verts[TRI_IDX(tid)], &verts.data[TRI_IDX(cur_idx)], sizeof(Vec4) * 3, pipe);
            // This memory op is not fully coalesced, since AABB container is not a complete SOA
            s_aabbs[tid].aabb.copy_from(aabbs[cur_idx]);
        }
        pipe.producer_commit();
        pipe.consumer_wait();
        __syncthreads();
        // this might not be a good solution
       ray.hit_t = ray_intersect(s_verts_arr, ray, s_aabbs, remain_prims, 
                cp_base << BASE_SHFL, min_index, min_object_id, prim_u, prim_v, ray.hit_t);
        __syncthreads();
    }
#endif  // RENDERER_USE_BVH

    // ============= step 2: local shading for indirect bounces ================
    if (block_index < num_valid && min_index >= 0) {
        // if the ray hits nothing, or the path throughput is 0, then the ray will be inactive
        // inactive rays will only be processed in the miss_shader
        ray.set_hit();
        ray.set_hit_index(min_index);
#ifdef FUSED_MISS_SHADER
        ray.set_active(true);
#endif   // FUSED_MISS_SHADER
        it.it() = Primitive::get_interaction(verts, *norms, *uvs, ray.advance(ray.hit_t), prim_u, prim_v, min_index, min_object_id >= 0);
    }

    payloads.set_ray(px, py, ray);
    payloads.set_interaction(px, py, it);
}

/***
 * For non-delta hit (shading point), direct component should be evaluated:
 * we sample a light source then start ray intersection test
*/
CPT_KERNEL void nee_shader(
    const PrecomputedArray& verts,
    PayLoadBufferSoA payloads,
    ConstObjPtr objects,
    ConstAABBPtr aabbs,
    ConstNormPtr norms, 
    ConstUVPtr,         
    const hipTextureObject_t bvh_leaves,
    const hipTextureObject_t node_fronts,
    const hipTextureObject_t node_backs,
    ConstF4Ptr cached_nodes,
    const IndexBuffer idx_buffer,
    int stream_offset,
    int num_prims,
    int num_objects,
    int num_emitter,
    int num_valid,
    int node_num,
    int cache_num
) {
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px
#ifdef RENDERER_USE_BVH
    // cache near root level BVH nodes for faster traversal
    extern __shared__ float4 s_cached[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < 2 * cache_num) {      // no more than 128 nodes will be cached
        s_cached[tid] = cached_nodes[tid];
    }
    __syncthreads();
#endif  // RENDERER_USE_BVH
    
    if (block_index < num_valid) {
        uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
        py >>= 16;
        Vec4 thp = payloads.thp(px, py);
        Ray ray  = payloads.get_ray(px, py);
        Sampler sg = payloads.get_sampler(px, py);
        const PDFInteraction it = payloads.get_interaction(px, py);

        auto aabb_front = CONST_FLOAT4(aabbs[ray.hit_id()].mini);       // hope to have coalesced access
        int object_id   = __float_as_int(aabb_front.w),
            material_id = objects[object_id].bsdf_id,
            emitter_id  = objects[object_id].emitter_id;

        float direct_pdf = 1;

        Emitter* emitter = sample_emitter(sg, direct_pdf, num_emitter, emitter_id);
        emitter_id       = objects[emitter->get_obj_ref()].sample_emitter_primitive(sg.discrete1D(), direct_pdf);
        Ray shadow_ray(ray.advance(ray.hit_t), Vec3(0, 0, 0));
        // use ray.o to avoid creating another shadow_int variable
        Vec4 direct_comp(0, 0, 0, 1);
        shadow_ray.d = emitter->sample(shadow_ray.o, direct_comp, direct_pdf, sg.next2D(), &verts, norms, emitter_id) - shadow_ray.o;

        float emit_len_mis = shadow_ray.d.length();
        shadow_ray.d *= __frcp_rn(emit_len_mis);              // normalized direct
        // (3) NEE scene intersection test (possible warp divergence, but... nevermind)
        if (emitter != c_emitter[0] && 
#ifdef RENDERER_USE_BVH
            occlusion_test_bvh(shadow_ray, bvh_leaves, node_fronts, node_backs, 
                    s_cached, verts, node_num, cache_num, emit_len_mis - EPSILON)
#else   // RENDERER_USE_BVH
            occlusion_test(shadow_ray, objects, aabbs, verts, num_objects, emit_len_mis - EPSILON)
#endif  // RENDERER_USE_BVH
        ) {
            // MIS for BSDF / light sampling, to achieve better rendering
            // 1 / (direct + ...) is mis_weight direct_pdf / (direct_pdf + material_pdf), divided by direct_pdf
            emit_len_mis = direct_pdf + c_material[material_id]->pdf(it.it_const(), shadow_ray.d, ray.d) * emitter->non_delta();
            payloads.L(px, py) += thp * direct_comp * c_material[material_id]->eval(it.it_const(), shadow_ray.d, ray.d) * \
                (float(emit_len_mis > EPSILON) * __frcp_rn(emit_len_mis < EPSILON ? 1.f : emit_len_mis));
            // numerical guard, in case emit_len_mis is 0
        }

        payloads.set_sampler(px, py, sg);
    }
}

/**
 * BSDF sampling & direct shading shader
*/
CPT_KERNEL void bsdf_local_shader(
    PayLoadBufferSoA payloads,
    ConstObjPtr objects,
    ConstAABBPtr aabbs,
    ConstUVPtr,         
    const IndexBuffer idx_buffer,
    int stream_offset,
    int num_prims, 
    int num_valid,
    bool secondary_bounce
) {
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px

    if (block_index < num_valid) {
        uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
        py >>= 16;

        Vec4 thp = payloads.thp(px, py);
        Ray ray  = payloads.get_ray(px, py);
        Sampler sg = payloads.get_sampler(px, py);
        PDFInteraction it = payloads.get_interaction(px, py);
        Vec2 sample = sg.next2D();
        payloads.set_sampler(px, py, sg);

        auto aabb_front = CONST_FLOAT4(aabbs[ray.hit_id()].mini);       // hope to have coalesced access
        int object_id   = __float_as_int(aabb_front.w),
            emitter_id  = objects[object_id].emitter_id,
            material_id = objects[object_id].bsdf_id;
        bool hit_emitter = emitter_id > 0;

        // emitter MIS
        float emission_weight = it.pdf_v() / (it.pdf_v() + 
                objects[object_id].solid_angle_pdf(it.it_const().shading_norm, ray.d, ray.hit_t) * hit_emitter * secondary_bounce);
        // (2) check if the ray hits an emitter
        Vec4 direct_comp = thp *\
                    c_emitter[emitter_id]->eval_le(&ray.d, &it.it_const().shading_norm);
        payloads.L(px, py) += direct_comp * emission_weight;
        
        ray.o = ray.advance(ray.hit_t);
        ray.d = c_material[material_id]->sample_dir(
            ray.d, it.it_const(), thp, it.pdf(), std::move(sample)
        );

        payloads.thp(px, py) = thp;
        payloads.set_ray(px, py, ray);
        payloads.set_it_head(px, py, it.data.p1);
    }
}

/**
 * Purpose of the miss shader: if ray hits nothing in closesthit shader
 * the we will set the hit status (flag) to be false
 * in this shader, we find the rays marked as no-hit, and check the
 * availability of environment map (currently not supported)
 * after processing the env-map lighting, we mark the ray as inactive
 * before stream compaction. Then stream compaction will 'remove' all these
 * rays (and the threads)
 * 
 * MISS_SHADER is the only place where you mark a ray as inactive
*/
CPT_KERNEL void miss_shader(
    PayLoadBufferSoA payloads,
    const IndexBuffer idx_buffer,
    int stream_offset,
    int num_valid
) {
    // Nothing here, currently, if we decide not to support env lighting
    const int block_index = (threadIdx.y + blockIdx.y * blockDim.y) *           // py
                            blockDim.x * gridDim.x +                            // cols
                            threadIdx.x + blockIdx.x * blockDim.x;              // px
    if (block_index < num_valid) {
        uint32_t py = idx_buffer[block_index + stream_offset], px = py & 0x0000ffff;
        py >>= 16;
        Vec4 thp = payloads.thp(px, py);
        if ((!payloads.is_hit(px, py)) || thp.max_elem() <= 1e-5f) {
            // TODO: process no-hit ray, environment map lighting
            payloads.set_active(px, py, false);
        }
    }
}

template <bool render_once>
CPT_KERNEL void radiance_splat(
    PayLoadBufferSoA payloads, DeviceImage image, 
    int stream_id, int x_patch, int y_patch, 
    int accum_cnt, float* output_buffer
) {
    // Nothing here, currently, if we decide not to support env lighting
    const int px = threadIdx.x + blockIdx.x * blockDim.x, py = threadIdx.y + blockIdx.y * blockDim.y;
    Vec4 L = payloads.L(px + stream_id * PATCH_X, py);         // To local register
    L = L.numeric_err() ? Vec4(0, 0, 0, 1) : L;

    if constexpr (render_once) {
        // image will be the output buffer, there will be double buffering
        int img_x = px + x_patch * PATCH_X, img_y = py + y_patch * PATCH_Y;
        auto local_v = image(img_x, img_y) + L;
        image(img_x, img_y) = local_v;
        local_v *= 1.f / float(accum_cnt);
        FLOAT4(output_buffer[(img_x + img_y * image.w()) << 2]) = float4(local_v); 
    } else {
        image(px + x_patch * PATCH_X, py + y_patch * PATCH_Y) += L;
    }
}

template CPT_KERNEL void radiance_splat<true>(
    PayLoadBufferSoA payloads, DeviceImage image, 
    int stream_id, int x_patch, int y_patch,
    int accum_cnt, float* output_buffer
);

template CPT_KERNEL void radiance_splat<false>(
    PayLoadBufferSoA payloads, DeviceImage image, 
    int stream_id, int x_patch, int y_patch, 
    int accum_cnt, float* output_buffer
);