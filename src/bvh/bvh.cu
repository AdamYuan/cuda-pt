#include "hip/hip_runtime.h"
/**
 * @file bvh.cpp
 * @author Qianyue He
 * @date 2023.5 -> 2024.9
 * @brief BVH construction main logic
 * @copyright Copyright (c) 2023-2024
 */

#include <algorithm>
#include <numeric>
#include <array>
#include "bvh.cuh"

using IntPair = std::pair<int, bool>;

static constexpr int num_bins = 12;
static constexpr float traverse_cost = 0.4;
static constexpr float max_node_prim = 1;

SplitAxis BVHNode::max_extent_axis(const std::vector<BVHInfo>& bvhs, std::vector<float>& bins) const {
    Vec3 min_ctr = bvhs[base].centroid, max_ctr = bvhs[base].centroid;
    for (int i = 1; i < prim_num; i++) {
        Vec3 ctr = bvhs[base + i].centroid;
        min_ctr.minimized(ctr);
        max_ctr.maximized(ctr);
    }
    Vec3 diff = max_ctr - min_ctr;
    float max_diff = diff.x();
    int split_axis = 0;
    if (diff.y() > max_diff) {
        max_diff = diff.y();
        split_axis = 1;
    }
    if (diff.z() > max_diff) {
        max_diff = diff.z();
        split_axis = 2;
    }
    bins.resize(num_bins);
    float min_r = min_ctr[split_axis] - 0.001f, interval = (max_diff + 0.002f) / float(num_bins);
    std::transform(bins.begin(), bins.end(), bins.begin(), [min_r, interval, i = 0] (const float&) mutable {
        i++; return min_r + interval * float(i);
    });
    return SplitAxis(split_axis);
}

void index_input(
    const std::vector<ObjInfo>& objs, 
    const std::vector<bool>& sphere_flags, 
    std::vector<IntPair>& idxs, size_t num_primitives
) {
    // input follow the shape of the number of objects, for each position
    // the number of primitive / whether the primitive is sphere will be stored, the index will be object id
    size_t result_shape = objs.size();      // shape is (3, obj_num)
    idxs.reserve(num_primitives);                   // accumulate(num_ptr, num_ptr + result_shape) = num_primitives
    for (size_t i = 0; i < result_shape; i++) {
        const int prim_num = objs[i].prim_num;
        const bool sphere_flag = sphere_flags[i];
        for (int j = 0; j < prim_num; j++)
            idxs.emplace_back(static_cast<int>(i), sphere_flag);
    }
}

void create_bvh_info(
    const std::vector<Vec3>& points1,
    const std::vector<Vec3>& points2,
    const std::vector<Vec3>& points3,
    const std::vector<IntPair>& idxs, std::vector<BVHInfo>& bvh_infos) {
    bvh_infos.reserve(points1.size());
    printf("Point1 size: %lu, idx pair size: %lu\n", points1.size(), points2.size());
    for (size_t i = 0; i < points1.size(); i++) {
        const IntPair& idx_info = idxs[i];
        bvh_infos.emplace_back(points1[i], points2[i], points3[i], idx_info.first, idx_info.second);
    }
}

int recursive_bvh_SAH(BVHNode* const cur_node, std::vector<BVHInfo>& bvh_infos, int depth = 0) {
    AABB fwd_bound, bwd_bound;
    int child_prim_cnt = 0;                // this index is used for indexing variable `bins`
    const int prim_num = cur_node->prim_num, base = cur_node->base, max_pos = base + prim_num;
    float min_cost = 5e9, node_prim_cnt = float(prim_num), node_inv_area = 1. / cur_node->bound.area();

    // Step 1: decide the axis that expands the maximum extent of space
    std::vector<float> bins;        // bins: from (start_pos + interval) to end_pos
    SplitAxis max_axis = cur_node->max_extent_axis(bvh_infos, bins);
    printf("Max extend axis: %d, %d\n", max_axis, prim_num);
    if (cur_node->prim_num > 4) {   // SAH

        // Step 2: binning the space
        std::array<AxisBins, num_bins> idx_bins;
        for (int i = cur_node->base; i < max_pos; i++) {
            size_t index = std::lower_bound(bins.begin(), bins.end(), bvh_infos[i].centroid[max_axis]) - bins.begin();
            idx_bins[index].push(bvh_infos[i]);
        }

        // Step 3: forward-backward linear sweep for heuristic calculation
        std::array<int, num_bins> prim_cnts;
        std::array<float, num_bins> fwd_areas, bwd_areas;
        for (int i = 0; i < num_bins; i++) {
            fwd_bound   += idx_bins[i].bound;
            prim_cnts[i] = idx_bins[i].prim_cnt;
            fwd_areas[i] = fwd_bound.area();
            if (i > 0) {
                bwd_bound += idx_bins[num_bins - i].bound;
                bwd_areas[num_bins - 1 - i] = bwd_bound.area();
            }
        }
        std::partial_sum(prim_cnts.begin(), prim_cnts.end(), prim_cnts.begin());

        // Step 4: use the calculated area to computed the segment boundary
        int seg_bin_idx = 0;
        for (int i = 0; i < num_bins - 1; i++) {
            float cost = traverse_cost + node_inv_area * 
                (float(prim_cnts[i]) * fwd_areas[i] + (node_prim_cnt - (prim_cnts[i])) * bwd_areas[i]);
            if (cost < min_cost) {
                min_cost = cost;
                seg_bin_idx = i;
            }
        }
        // Step 5: reordering the BVH info in the vector to make the segment contiguous (partition around pivot)
        if (min_cost < node_prim_cnt) {
            std::partition(bvh_infos.begin() + base, bvh_infos.begin() + max_pos,
                [pivot = bins[seg_bin_idx], dim = max_axis](const BVHInfo& bvh) {
                    return bvh.centroid[dim] < pivot;
            });
            child_prim_cnt = prim_cnts[seg_bin_idx];
        }
        fwd_bound.clear();
        bwd_bound.clear();
        for (int i = 0; i <= seg_bin_idx; i++)       // calculate child node bound
            fwd_bound += idx_bins[i].bound;
        for (int i = num_bins - 1; i > seg_bin_idx; i--)
            bwd_bound += idx_bins[i].bound;
    } else {                                    // equal primitive number split (two nodes have identical primitives)
        int seg_idx = (base + max_pos) >> 1;
        // Step 5: reordering the BVH info in the vector to make the segment contiguous (keep around half of the bvh in lchild)
        std::nth_element(bvh_infos.begin() + base, bvh_infos.begin() + seg_idx, bvh_infos.begin() + max_pos,
            [dim = max_axis] (const BVHInfo& bvh1, const BVHInfo& bvh2) {
                return bvh1.centroid[dim] < bvh2.centroid[dim];
            }
        );
        for (int i = base; i < seg_idx; i++)    // calculate child node bound
            fwd_bound += bvh_infos[i].bound;
        for (int i = seg_idx; i < max_pos; i++)
            bwd_bound += bvh_infos[i].bound;
        child_prim_cnt = seg_idx - base;        // bvh[seg_idx] will be in rchild
        float split_cost = traverse_cost + node_inv_area * 
                (fwd_bound.area() * child_prim_cnt + bwd_bound.area() * (node_prim_cnt - child_prim_cnt));
        if (split_cost >= node_prim_cnt)
            child_prim_cnt = 0;
    }
    if (child_prim_cnt > 0) {             // cost of splitting is less than making this node a leaf node
        // Step 5: split the node and initialize the children
        cur_node->lchild = new BVHNode(base, child_prim_cnt);
        cur_node->rchild = new BVHNode(base + child_prim_cnt, prim_num - child_prim_cnt);

        cur_node->lchild->bound = fwd_bound;
        cur_node->rchild->bound = bwd_bound;
        cur_node->axis = max_axis;
        // Step 7: start recursive splitting for the children
        int node_num = 1;
        // if (depth == 0) {
        //     int local_node_n1 = 0, local_node_n2 = 0;
        //     #pragma omp parallel sections       // parallel SAH-BVH
        //     {
        //         #pragma omp section 
        //         {
        //             if (cur_node->lchild->prim_num > max_node_prim)
        //                 local_node_n1 = recursive_bvh_SAH(cur_node->lchild, bvh_infos, depth + 1);
        //             else local_node_n1 = 1;
        //         }
        //         #pragma omp section 
        //         {
        //             if (cur_node->rchild->prim_num > max_node_prim)
        //                 local_node_n2 = recursive_bvh_SAH(cur_node->rchild, bvh_infos, depth + 1);
        //             else local_node_n2 = 1;
        //         }
        //     }
        //     node_num = local_node_n1 + local_node_n2 + 1;
        // } else {
            if (cur_node->lchild->prim_num > max_node_prim)
                node_num += recursive_bvh_SAH(cur_node->lchild, bvh_infos, depth + 1);
            else node_num ++;
            if (cur_node->rchild->prim_num > max_node_prim)
                node_num += recursive_bvh_SAH(cur_node->rchild, bvh_infos, depth + 1);
            else node_num ++;
        // }
        return node_num;
    } else {
        // This is a leaf node, yet this is the only way that a leaf node contains more than one primitive
        cur_node->axis = AXIS_NONE;
        return 1;
    }
}

static BVHNode* bvh_root_start(const Vec3& world_min, const Vec3& world_max, int& node_num, std::vector<BVHInfo>& bvh_infos) {
    // Build BVH tree root node and start recursive tree construction
    printf("World min: ");
    print_vec3(world_min);
    printf("World max: ");
    print_vec3(world_max);
    BVHNode* root_node = new BVHNode(0, bvh_infos.size());
    root_node->bound.mini = world_min;
    root_node->bound.maxi = world_max;
    node_num = recursive_bvh_SAH(root_node, bvh_infos);
    return root_node;
}

// This is the final function call for `bvh_build`
static int recursive_linearize(BVHNode* cur_node, std::vector<LinearNode>& lin_nodes) {
    // BVH tree should be linearized to better traverse and fit in the system memory
    // The linearized BVH tree should contain: bound, base, prim_cnt, rchild_offset, total_offset (to skip the entire node)
    // Note that if rchild_offset is -1, then the node is leaf. Leaf node points to primitive array
    // which is already sorted during BVH construction, containing primitive_id and obj_id for true intersection
    // Note that lin_nodes has been reserved
    size_t current_size = lin_nodes.size();
    lin_nodes.emplace_back(cur_node);
    if (cur_node->lchild != nullptr) {
        // TODO: parallel linearize
        int lnodes = recursive_linearize(cur_node->lchild, lin_nodes);
        lnodes += recursive_linearize(cur_node->rchild, lin_nodes);
        lin_nodes[current_size].all_offset = lnodes + 1;
        return lnodes + 1;                      // include the cur_node                       
    } else {
        printf("Lin node size: %lu. Recursive End.\n", lin_nodes.size());
        lin_nodes.back().all_offset = 1;        // to skip the current sub-tree, index should just add 1
        return 1;
    }
}

// Try to use two threads to build the BVH
void bvh_build(
    const std::vector<Vec3>& points1,
    const std::vector<Vec3>& points2,
    const std::vector<Vec3>& points3,
    const std::vector<ObjInfo>& objects,
    const std::vector<bool>& sphere_flags,
    const Vec3& world_min, const Vec3& world_max,
    std::vector<LinearBVH>& lin_bvhs, std::vector<LinearNode>& lin_nodes
) {
    std::vector<IntPair> idx_prs;
    std::vector<BVHInfo> bvh_infos;
    int node_num = 0, num_prims_all = points1.size();
    index_input(objects, sphere_flags, idx_prs, num_prims_all);
    create_bvh_info(points1, points2, points3, idx_prs, bvh_infos);
    BVHNode* root_node = bvh_root_start(world_min, world_max, node_num, bvh_infos);
    recursive_linearize(root_node, lin_nodes);
    lin_bvhs.reserve(bvh_infos.size());
    for (const BVHInfo& bvh: bvh_infos) {
        lin_bvhs.emplace_back(bvh);
    }
    delete root_node;
}