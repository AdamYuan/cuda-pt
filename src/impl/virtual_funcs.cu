/**
 * Utilization function to copy the classes with virtual functions to GPU
 * Implementation
 * @author: Qianyue He
 * @date:   2024.9.6
*/

#include "core/virtual_funcs.cuh"

CPT_KERNEL void create_point_source(Emitter* &dst, Vec4 le, Vec3 pos) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        dst = new PointSource(std::move(le), std::move(pos));
    }
}

CPT_KERNEL void create_area_source(Emitter* &dst, Vec4 le, int obj_ref, bool is_sphere, hipTextureObject_t obj) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        dst = new AreaSource(std::move(le), obj_ref, is_sphere, obj);
    }
}

CPT_KERNEL void create_area_spot_source(Emitter* &dst, Vec4 le, float cos_val, int obj_ref, bool is_sphere, hipTextureObject_t obj) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        dst = new AreaSpotSource(std::move(le), cos_val, obj_ref, is_sphere, obj);
    }
}

CPT_KERNEL void create_envmap_source(Emitter* &dst, hipTextureObject_t obj, float scaler, float azimuth, float zenith) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        dst = new EnvMapEmitter(obj, scaler, azimuth, zenith);
        dst->set_func1(scaler);
        dst->set_func2(azimuth);
        dst->set_func3(zenith);
    }
}

CPT_KERNEL void set_emission(Emitter* &dst, Vec3 color, float scaler) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        dst->set_le(std::move(color), scaler);
    }
}

CPT_KERNEL void call_setter(Emitter* &dst, float v1, float v2, float v3) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        dst->set_func1(v1);
        dst->set_func2(v2);
        dst->set_func3(v3);
    }
}

CPT_KERNEL void create_abstract_source(Emitter* &dst) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        dst = new Emitter(Vec4(0, 0, 0));
    }
}

CPT_KERNEL void create_metal_bsdf(BSDF** dst, Vec3 eta_t, Vec3 k, Vec4 k_g, float roughness_x, float roughness_y) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        if (*dst) delete *dst;
        *dst = new GGXConductorBSDF(eta_t, k, k_g, roughness_x, roughness_y);
    }
}

CPT_KERNEL void load_metal_bsdf(
    BSDF** dst, Vec3 eta_t, Vec3 k, Vec4 k_g, float roughness_x, float roughness_y
) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // I will make sure (I can) the base ptr is actually of PlasticType*
        // So dynamic_cast is not needed (actually, not allowed on device code)
        GGXConductorBSDF* ptr = static_cast<GGXConductorBSDF*>(*dst);
        ptr->fresnel = FresnelTerms(std::move(eta_t), std::move(k));
        ptr->set_kd(Vec4(0));
        ptr->set_ks(Vec4(roughness_to_alpha(roughness_x), roughness_to_alpha(roughness_y), 1));
        ptr->set_kg(std::move(k_g));
    }
}

CPT_KERNEL void load_dispersion_bsdf(
    BSDF** dst, Vec4 k_s, float index_a, float index_b
) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*dst)->set_kd(Vec4(index_a, index_b, 0));
        (*dst)->set_ks(std::move(k_s));
    }
}

CPT_KERNEL void create_dispersion_bsdf(
    BSDF** dst, Vec4 k_s, float index_a, float index_b
) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        if (*dst) delete *dst;
        *dst = new DispersionBSDF(k_s, index_a, index_b);
        (*dst)->set_kd(Vec4(index_a, index_b, 0));
        (*dst)->set_ks(std::move(k_s));
    }
}

CPT_KERNEL void create_forward_bsdf(
    BSDF** dst
) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        if (*dst) delete *dst;
        *dst = new ForwardBSDF(ScatterStateFlag::BSDF_NONE);
    }
}

