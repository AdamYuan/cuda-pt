#include "hip/hip_runtime.h"
/**
 * Scene parser (from xml)
 * This is the implementation file
 * @author: Qianyue He
 * @date:   2024.9.6
*/
#include <numeric>
#include "core/scene.cuh"

static constexpr int MAX_PRIMITIVE_NUM = 64000000;
static constexpr int MAX_ALLOWED_BSDF = 48;
static constexpr const char* SCENE_VERSION = "1.2";

const std::unordered_map<std::string, MetalType> conductor_mapping = {
    {"Au", MetalType::Au},
    {"Cr", MetalType::Cr},
    {"Cu", MetalType::Cu},
    {"Ag", MetalType::Ag},
    {"Al", MetalType::Al},
    {"W",   MetalType::W},
    {"TiO2", MetalType::TiO2},
    {"Ni",  MetalType::Ni},
    {"MgO", MetalType::MgO},
    {"Na",  MetalType::Na},
    {"SiC", MetalType::SiC},
    {"V",   MetalType::V},
    {"CuO", MetalType::CuO},
    {"Hg",  MetalType::Hg},
    {"Ir",  MetalType::Ir},
};

const std::unordered_map<std::string, DispersionType> dielectric_mapping = {
    {"Diamond",     DispersionType::Diamond},
    {"DiamondHigh", DispersionType::DiamondHigh},
    {"Silica",      DispersionType::Silica},
    {"Glass_BK7",   DispersionType::Glass_BK7},
    {"Glass_BaF10", DispersionType::Glass_BaF10},
    {"Glass_SF10",  DispersionType::Glass_SF10},
    {"Sapphire",    DispersionType::Sapphire},
    {"Water",       DispersionType::Water}
};

static std::string get_folder_path(std::string filePath) {
    size_t pos = filePath.find_last_of("/\\");
    if (pos != std::string::npos) {
        return filePath.substr(0, pos + 1); // includes the last '/'
    }
    return ""; // include empty str if depth is 0
}

Vec4 parseColor(const std::string& value) {
    float r, g, b;
    if (value[0] == '#') {
        std::stringstream ss;
        ss << std::hex << value.substr(1);
        unsigned int color;
        ss >> color;
        r = float((color >> 16) & 0xFF) / 255.0f;
        g = float((color >> 8) & 0xFF) / 255.0f;
        b = float(color & 0xFF) / 255.0f;
    } else if (value.find(',') != std::string::npos || value.find(' ') != std::string::npos) {
        std::stringstream ss(value);
        std::vector<float> values;
        float component;
        while (ss >> component) {
            values.push_back(component);
            if (ss.peek() == ',' || ss.peek() == ' ') {
                ss.ignore();
            }
        }
        r = values[0];
        g = values[1];
        b = values[2];
    } else {
        std::stringstream ss(value);
        ss >> r;
        g = r;
        b = r;
    }
    return Vec4(r, g, b);
}

Vec3 parsePoint(const tinyxml2::XMLElement* element) {
    if (element == nullptr) {
        std::cerr << "Point not specified for point source.\n";
        throw std::runtime_error("Point element is null");
    }

    const char* name = element->Attribute("name");
    if (name == nullptr) {
        throw std::runtime_error("No 'name' attribute found");
    }

    float x = 0, y = 0, z = 0;
    tinyxml2::XMLError eResult = element->QueryFloatAttribute("x", &x);
    if (eResult != tinyxml2::XML_SUCCESS) {
        throw std::runtime_error("Error parsing 'x' attribute");
    }

    eResult = element->QueryFloatAttribute("y", &y);
    if (eResult != tinyxml2::XML_SUCCESS) {
        throw std::runtime_error("Error parsing 'y' attribute");
    }

    eResult = element->QueryFloatAttribute("z", &z);
    if (eResult != tinyxml2::XML_SUCCESS) {
        throw std::runtime_error("Error parsing 'z' attribute");
    }

    return Vec3(x, y, z);
}

void parseBSDF(
    const tinyxml2::XMLElement* bsdf_elem, 
    const std::unordered_map<std::string, TextureInfo>& tex_map,
    std::unordered_map<std::string, int>& bsdf_map, 
    std::vector<BSDFInfo>& bsdf_infos,
    std::vector<Texture<float4>>& host_4d,
    std::vector<Texture<float2>>& host_2d,
    Textures& textures,
    BSDF** bsdfs, 
    int index
) {
    std::string type = bsdf_elem->Attribute("type");
    std::string id = bsdf_elem->Attribute("id");

    bsdf_map[id] = index;
    Vec4 k_d, k_s, k_g;

    const tinyxml2::XMLElement* element = bsdf_elem->FirstChildElement("rgb");
    while (element) {
        std::string name = element->Attribute("name");
        std::string value = element->Attribute("value");
        Vec4 color = parseColor(value);
        if (name == "k_d") {
            k_d = color;
        } else if (name == "k_s") {
            k_s = color;
        } else if (name == "k_g" || name == "sigma_a") {
            k_g = color;
        }
        element = element->NextSiblingElement("rgb");
    }

    // reference to the texture
    element = bsdf_elem->FirstChildElement("ref");
    if (element) {
        std::string name = element->Attribute("type");
        if (!name.empty() && name == "texture") {
            std::string value = element->Attribute("id");
            auto it = tex_map.find(value);
            if (it == tex_map.end()) {
                std::cerr << "Texture named '" << value  << "' not found.\n";
                throw std::runtime_error("Referenced Texture not found.");
            } else {
                if (!it->second.diff_path.empty()) {
                    Texture<float4> tex(it->second.diff_path, TextureType::DIFFUSE_TEX);
                    textures.enqueue(tex, index);
                    host_4d.emplace_back(std::move(tex));
                }
                if (!it->second.spec_path.empty()) {
                    Texture<float4> tex(it->second.spec_path, TextureType::SPECULAR_TEX);
                    textures.enqueue(tex, index);
                    host_4d.emplace_back(std::move(tex));
                }
                if (!it->second.glos_path.empty()) {
                    Texture<float4> tex(it->second.glos_path, TextureType::GLOSSY_TEX);
                    textures.enqueue(tex, index);
                    host_4d.emplace_back(std::move(tex));
                }
                if (!it->second.rough_path1.empty()) {
                    Texture<float2> tex(
                        it->second.rough_path1, 
                        TextureType::ROUGHNESS_TEX, 
                        it->second.rough_path2, 
                        it->second.is_rough_ior
                    );
                    textures.enqueue(tex, index);
                    host_2d.emplace_back(std::move(tex));
                }
                if (!it->second.normal_path.empty()) {
                    Texture<float4> tex(it->second.normal_path, TextureType::NORMAL_TEX, "", false, true);
                    textures.enqueue(tex, index);
                    host_4d.emplace_back(std::move(tex));
                }
            }   
        }
    }

    BSDFInfo info(id);
    info.bsdf = BSDFInfo::BSDFParams(k_d, k_s, k_g);
    if (type == "lambertian") {
        create_bsdf<LambertianBSDF><<<1, 1>>>(bsdfs + index, k_d, k_s, k_g, ScatterStateFlag::BSDF_DIFFUSE | ScatterStateFlag::BSDF_REFLECT);
    } else if (type == "specular") {
        create_bsdf<SpecularBSDF><<<1, 1>>>(bsdfs + index, k_d, k_s, k_g, ScatterStateFlag::BSDF_SPECULAR | ScatterStateFlag::BSDF_REFLECT);
        info.type = BSDFType::Specular;
    } else if (type == "det-refraction") {
        create_bsdf<TranslucentBSDF><<<1, 1>>>(bsdfs + index, k_d, k_s, k_g, ScatterStateFlag::BSDF_SPECULAR | ScatterStateFlag::BSDF_TRANSMIT);
        info.type = BSDFType::Translucent;
    } else if (type == "conductor-ggx") {
        float roughness_x = 0.1f, roughness_y = 0.1f;
        MetalType mtype = MetalType::Cu;
        info.type = BSDFType::GGXConductor;
        element = bsdf_elem->FirstChildElement("string");
        if (element) {
            std::string name = element->Attribute("name");
            std::string value = element->Attribute("value");
            if (name == "type" || name == "metal" || name == "conductor") {
                std::string metal_type = element->Attribute("value");
                auto it = conductor_mapping.find(metal_type);
                if (it == conductor_mapping.end()) {
                    std::cout << "BSDF[" << id << "]" << ": Only << " << int(NumMetalType) << " types of metals are supported: ";
                    for (const auto [k, v]: conductor_mapping)
                        std::cout << k << ", ";
                    std::cout << std::endl;
                    std::cout << "Current type '" << metal_type << "' is not supported. Setting to 'Cu'\n";
                } else {
                    mtype = it->second;
                }
            }
        }
        element = bsdf_elem->FirstChildElement("float");
        tinyxml2::XMLError eResult;
        while (element) {
            std::string name = element->Attribute("name");
            std::string value = element->Attribute("value");
            if (name == "roughness_x" || name == "rough_x") {
                eResult = element->QueryFloatAttribute("value", &roughness_x);
                roughness_x = std::clamp(roughness_x, 0.001f, 1.f);
            } else if (name == "roughness_y" || name == "rough_y") {
                eResult = element->QueryFloatAttribute("value", &roughness_y);
                roughness_y = std::clamp(roughness_y, 0.001f, 1.f);
            }
            if (eResult != tinyxml2::XML_SUCCESS)
                throw std::runtime_error("Error parsing 'roughness' attribute");
            element = element->NextSiblingElement("float");
        }
        info.bsdf.store_ggx_params(mtype, k_g, roughness_x, roughness_y);
        create_metal_bsdf<<<1, 1>>>(bsdfs + index, METAL_ETA_TS[mtype], 
                    METAL_KS[mtype], k_g, roughness_x, roughness_y);
    } else if (type == "plastic" || type == "plastic-forward") {
        k_g = Vec4(0, 1);
        element = bsdf_elem->FirstChildElement("float");
        float trans_scaler = 1.f, thickness = 0.f, ior = 1.33f;
        bool penetrable = false;
        while (element) {
            std::string name = element->Attribute("name");
            tinyxml2::XMLError eResult;
            if (name == "trans_scaler") {
                eResult = element->QueryFloatAttribute("value", &trans_scaler);
            } else if (name == "thickness") {
                eResult = element->QueryFloatAttribute("value", &thickness);
            } else if (name == "ior") {
                eResult = element->QueryFloatAttribute("value", &ior);
            }
            if (eResult != tinyxml2::XML_SUCCESS)
                throw std::runtime_error("Error parsing 'plastic BRDF' attribute");
            element = element->NextSiblingElement("float");
        }
        element = bsdf_elem->FirstChildElement("bool");
        if (element) {
            if (std::string(element->Attribute("name")) == "penetrable") {
                auto eResult = element->QueryBoolAttribute("value", &penetrable);
                if (eResult != tinyxml2::XML_SUCCESS)
                    throw std::runtime_error("Error parsing 'plastic BRDF' attribute");
            }
        }
        if (type == "plastic") {
            info.type = BSDFType::Plastic;
            create_plastic_bsdf<PlasticBSDF><<<1, 1>>>(bsdfs + index, 
                    k_d, k_s, k_g, ior, trans_scaler, thickness, penetrable);
        } else {
            info.type = BSDFType::PlasticForward;
            create_plastic_bsdf<PlasticForwardBSDF><<<1, 1>>>(bsdfs + index, 
                    k_d, k_s, k_g, ior, trans_scaler, thickness, penetrable);
        }
        info.bsdf.store_plastic_params(ior, trans_scaler, thickness);
    } else if (type == "dispersion") {
        element = bsdf_elem->FirstChildElement("string");
        DispersionType dtype = DispersionType::Diamond;
        if (element) {
            std::string name = element->Attribute("name");
            if (name == "type" || name == "dielectric") {
                std::string dielec_type = element->Attribute("value");
                auto it = dielectric_mapping.find(dielec_type);
                if (it == dielectric_mapping.end()) {
                    std::cout << "BSDF[" << id << "]" << ": Only 8 types of metals are supported: ";
                    for (const auto [k, v]: dielectric_mapping)
                        std::cout << k << ", ";
                    std::cout << std::endl;
                    std::cout << "Current type '" << dielec_type << "' is not supported. Setting to 'Diamond'\n";
                } else {
                    dtype = it->second;
                }
            }
        }
        Vec2 dis_params = DISPERSION_PARAMS[dtype];
        info.type = BSDFType::Dispersion;
        info.bsdf.store_dispersion_params(dtype, k_s);
        create_dispersion_bsdf<<<1, 1>>>(bsdfs + index, k_s, dis_params.x(), dis_params.y());
    }
    bsdf_infos.emplace_back(std::move(info));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

void parseEmitterNames(
    const tinyxml2::XMLElement* emitter_elem, 
    std::unordered_map<std::string, int>& emitter_map
) {
    int idx = 1;
    while (emitter_elem) {
        std::string id = emitter_elem->Attribute("id");
        emitter_map[id] = idx++;
        emitter_elem = emitter_elem->NextSiblingElement("emitter");
    }
}

void parseEmitter(
    const tinyxml2::XMLElement* emitter_elem, 
    std::unordered_map<std::string, int>& emitter_obj_map,      // key emitter name, value object_id,
    const std::unordered_map<std::string, TextureInfo>& tex_map,
    std::vector<std::pair<std::string, Vec4>>& e_props,
    std::vector<std::string> obj_ref_names,
    std::vector<Texture<float4>>& host_texs,
    Emitter** emitters, 
    int& envmap_id,
    int index
) {
    std::string type = emitter_elem->Attribute("type");
    std::string id = emitter_elem->Attribute("id");

    obj_ref_names.push_back(id);
    Vec4 emission(0, 0, 0), scaler(1, 1, 1);

    const tinyxml2::XMLElement* element = emitter_elem->FirstChildElement("rgb");
    while (element) {
        std::string name = element->Attribute("name");
        std::string value = element->Attribute("value");
        Vec4 color = parseColor(value);
        if (name == "emission") {
            emission = color;
        } else if (name == "scaler") {
            scaler = color;
        }
        element = element->NextSiblingElement("rgb");
    }
    scaler.w() = scaler.x();
    e_props.emplace_back(id, Vec4(emission.xyz(), scaler.x()));

    hipTextureObject_t tex_obj = 0;
    element = emitter_elem->FirstChildElement("ref");
    if (element) {
        std::string name = element->Attribute("type");
        if (!name.empty() && name == "texture") {
            std::string value = element->Attribute("id");
            auto it = tex_map.find(value);
            if (it == tex_map.end()) {
                std::cerr << "Texture named '" << value  << "' not found.\n";
                throw std::runtime_error("Referenced Texture not found.");
            } else {
                if (!it->second.diff_path.empty()) {
                    Texture<float4> tex(it->second.diff_path, TextureType::DIFFUSE_TEX);
                    tex_obj = tex.object();
                    host_texs.emplace_back(std::move(tex));
                } else {
                    std::cerr << "The texture for HDRI should be set for it's 'emission' element, but none is found.\n";
                    throw std::runtime_error("Referenced Texture not found.");
                }
            }
        }
    }

    if (type == "point") {
        element = emitter_elem->FirstChildElement("point");
        Vec3 pos(0, 0, 0);
        std::string name = element->Attribute("name");
        if (name == "center" || name == "pos")
            pos = parsePoint(element);
        create_point_source<<<1, 1>>>(emitters[index], emission * scaler, pos);
    } else if (type == "area-spot") {
        element = emitter_elem->FirstChildElement("float");
        float cos_val = 0.99;
        if (element) {
            std::string name = element->Attribute("name");
            tinyxml2::XMLError eResult;
            if (name == "half-angle" || name == "angle") {
                eResult = element->QueryFloatAttribute("value", &cos_val);
                cos_val = cosf(cos_val * DEG2RAD);
            }
            if (eResult != tinyxml2::XML_SUCCESS)
                throw std::runtime_error("Error parsing 'Area Spot Emitter' attribute");
        }
        element = emitter_elem->FirstChildElement("string");
        std::string attr_name = element->Attribute("name");
        if (!element || attr_name != "bind_type") {
            std::cerr << "Bound primitive is not specified for area spot source '" << id << "', name: "<< element->Attribute("name") << std::endl;
            throw std::runtime_error("Bound primitive is not specified for area spot source");
        }
        bool spherical_bound = element->Attribute("value") == std::string("sphere");
        create_area_spot_source<<<1, 1>>>(emitters[index], emission * scaler, cos_val, emitter_obj_map[id], spherical_bound, tex_obj);
    } else if (type == "area") {
        element = emitter_elem->FirstChildElement("string");
        std::string attr_name = element->Attribute("name");
        if (!element || attr_name != "bind_type") {
            std::cerr << "Bound primitive is not specified for area source '" << id << "', name: "<< element->Attribute("name") << std::endl;
            throw std::runtime_error("Bound primitive is not specified for area source");
        }
        bool spherical_bound = element->Attribute("value") == std::string("sphere");
        create_area_source<<<1, 1>>>(emitters[index], emission * scaler, emitter_obj_map[id], spherical_bound, tex_obj);
    } else if (type == "envmap") {
        envmap_id = index;
        element = emitter_elem->FirstChildElement("float");
        float scaler = 1.f, azimuth = 0.f, zenith = 0.f;
        while (element) {
            std::string name = element->Attribute("name");
            tinyxml2::XMLError eResult;
            if (name == "scaler") {
                eResult = element->QueryFloatAttribute("value", &scaler);
            } else if (name == "azimuth") {
                eResult = element->QueryFloatAttribute("value", &azimuth);
            } else if (name == "zenith") {
                eResult = element->QueryFloatAttribute("value", &zenith);
            }
            if (eResult != tinyxml2::XML_SUCCESS)
                throw std::runtime_error("Error parsing 'EnvMap Emitter' attribute");
            element = element->NextSiblingElement("float");
        }
        e_props.back().second = Vec4(-1, scaler, azimuth, zenith);
        element = emitter_elem->FirstChildElement("ref");
        if (tex_obj != 0) {
            create_envmap_source<<<1, 1>>>(emitters[index], 
                tex_obj, scaler, azimuth * DEG2RAD, zenith * DEG2RAD);
        } else {
            std::cerr << "Error: The texture for EnvMap is empty.\n";
            throw std::runtime_error("Referenced Texture not available.");
        }
    }
}

int get_map_id(const std::unordered_map<std::string, int>& map, const std::string& id) {
    auto it = map.find(id);
    if (it != map.end()) {
        return it->second;
    } else {
        std::cerr << "Map has no key: '" << id << "'\n";
        throw std::runtime_error("Map has no key: '" + id + "'");
    }
    return 0;
}

void parseSphereShape(
    const tinyxml2::XMLElement* shapeElement, 
    const std::unordered_map<std::string, int>& bsdf_map,
    const std::unordered_map<std::string, int>& emitter_map,
    std::vector<BSDFInfo>& bsdf_infos,
    std::unordered_map<std::string, int>& emitter_obj_map,
    std::vector<ObjInfo>& objects, std::array<Vec3Arr, 3>& verts_list, 
    std::array<Vec3Arr, 3>& norms_list, std::array<Vec2Arr, 3>& uvs_list, 
    int& prim_offset, std::string folder_prefix, int index
) {
    int bsdf_id = -1, emitter_id = 0;

    const tinyxml2::XMLElement* element = shapeElement->FirstChildElement("ref");
    
    while (element) {
        std::string type = element->Attribute("type");
        std::string id = element->Attribute("id");
        if (type == "material") {
            bsdf_id = get_map_id(bsdf_map, id);
        } else if (type == "emitter") {
            emitter_id = get_map_id(emitter_map, id);
            emitter_obj_map[id] = index;
        }
        element = element->NextSiblingElement("ref");
    }
    if (bsdf_id == -1) {
        std::cerr << "The current object does not have an attached BSDF.\n";
        throw std::runtime_error("Object with no BSDF");
    }
    bsdf_infos[bsdf_id].in_use = true;

    float radius = 0;
    Vec3 center(0, 0, 0);
    element = shapeElement->FirstChildElement("point");
    std::string name = element->Attribute("name");
    if (name == "center" || name == "pos")
        center = parsePoint(element);

    element = shapeElement->FirstChildElement("float");
    name = element->Attribute("name");
    if (name == "r" || name == "radius") {
        element->QueryFloatAttribute("value", &radius);
    }
    verts_list[0].emplace_back(std::move(center));
    verts_list[1].emplace_back(radius, radius, radius);
    verts_list[2].emplace_back(0, 0, 0);

    for (int i = 0; i < 3; i++) {
        norms_list[i].emplace_back(0, 1, 0);
        uvs_list[i].emplace_back(0, 0);
    }

    objects.emplace_back(bsdf_id, prim_offset, 1, emitter_id);
    objects.back().setup(verts_list, false);
    ++ prim_offset;
}

void parseObjShape(
    const tinyxml2::XMLElement* shapeElement, 
    const std::unordered_map<std::string, int>& bsdf_map,
    const std::unordered_map<std::string, int>& emitter_map,
    std::vector<BSDFInfo>& bsdf_infos,
    std::unordered_map<std::string, int>& emitter_obj_map,
    std::vector<ObjInfo>& objects, std::array<Vec3Arr, 3>& verts_list, 
    std::array<Vec3Arr, 3>& norms_list, std::array<Vec2Arr, 3>& uvs_list, 
    int& prim_offset, std::string folder_prefix, int index
) {
    std::string filename, name;
    int bsdf_id = -1, emitter_id = 0;

    const tinyxml2::XMLElement* element = shapeElement->FirstChildElement("string");
    while (element) {
        name = element->Attribute("name");
        if (name == "filename") {
            filename = folder_prefix + element->Attribute("value");
        }
        element = element->NextSiblingElement("string");
    }

    element = shapeElement->FirstChildElement("ref");
    while (element) {
        std::string type = element->Attribute("type");
        std::string id = element->Attribute("id");
        if (type == "material") {
            bsdf_id = get_map_id(bsdf_map, id);
        } else if (type == "emitter") {
            emitter_id = get_map_id(emitter_map, id);
            emitter_obj_map[id] = index;
        }
        element = element->NextSiblingElement("ref");
    }
    if (bsdf_id == -1) {
        std::cerr << "The current object does not have an attached BSDF.\n";
        throw std::runtime_error("Object with no BSDF");
    }
    bsdf_infos[bsdf_id].in_use = true;

    tinyobj::attrib_t attrib;
    std::vector<tinyobj::shape_t> shapes;
    std::vector<tinyobj::material_t> materials;
    std::string warn, err;

    bool ret = tinyobj::LoadObj(&attrib, &shapes, &materials, &warn, &err, filename.c_str());
    if (!ret) {
        std::cerr << "Failed to load/parse .obj file: " << filename << std::endl;
        return;
    }

    int num_new_primitive = 0;
    for (const auto& shape : shapes) {
        int new_vert_num = shape.mesh.indices.size();
        if (new_vert_num % 3)
            std::cerr << "Warning: the number of primitives is not divisible by 3.\n";
        num_new_primitive += new_vert_num / 3;
    }
    for (int i = 0; i < 3; i++) {
        verts_list[i].reserve(verts_list.size() + num_new_primitive);
        norms_list[i].reserve(norms_list.size() + num_new_primitive);
        uvs_list[i].reserve(uvs_list.size() + num_new_primitive);
    }
    ObjInfo object(bsdf_id, prim_offset, num_new_primitive, emitter_id);
    for (const auto& shape : shapes) {
        size_t num_primitives = shape.mesh.indices.size() / 3;
        prim_offset += num_primitives;

        for (size_t i = 0; i < num_primitives; ++i) {
            int prim_base = 3 * i;
            bool has_normal = false;
            for (int j = 0; j < 3; ++j) {
                const tinyobj::index_t& idx = shape.mesh.indices[prim_base + j];
                int index = idx.vertex_index * 3;
                verts_list[j].emplace_back(attrib.vertices[index], attrib.vertices[index + 1], attrib.vertices[index + 2]);

                if (idx.normal_index >= 0) {
                    has_normal = true;
                    index = 3 * idx.normal_index;
                    norms_list[j].emplace_back(attrib.normals[index], attrib.normals[index + 1], attrib.normals[index + 2]);
                }
                if (idx.texcoord_index >= 0) {
                    index = 2 * idx.texcoord_index;
                    uvs_list[j].emplace_back(attrib.texcoords[index], attrib.texcoords[index + 1]);
                } else {
                    uvs_list[j].emplace_back(0, 0);
                }
            }
            if (!has_normal) {      // compute normals ourselves
                printf("Normal vector not found in '%s' primitive %llu, computing yet normal direction is not guaranteed.\n", name.c_str(), i);
                Vec3 diff = verts_list[1][i] - verts_list[0][i];
                Vec3 normal = diff.cross(verts_list[2][i] - verts_list[0][i]).normalized_h();
                for (int j = 0; j < 3; j++) {
                    norms_list[j].push_back(normal);
                }
            }
        }

    }
    object.setup(verts_list);
    objects.push_back(object);
}

void parseTexture(
    const tinyxml2::XMLElement* tex_elem, 
    std::unordered_map<std::string, TextureInfo>& texs,
    std::string folder_prefix
) {
    while (tex_elem) {
        std::string id = tex_elem->Attribute("id");
        TextureInfo info;
        const tinyxml2::XMLElement* element = tex_elem->FirstChildElement("string");
        while (element) {
            std::string name = element->Attribute("name");
            if (name == "diffuse" || name == "emission") {
                info.diff_path = folder_prefix + element->Attribute("value");
            } else if (name == "specular") {
                info.spec_path = folder_prefix + element->Attribute("value");
            } else if (name == "glossy" || name == "sigma_a") {
                info.glos_path = folder_prefix + element->Attribute("value");
            } else if (name == "rough1" || name == "roughness_1" || name == "ior") {
                info.rough_path1 = folder_prefix + element->Attribute("value");
                info.is_rough_ior = name == "ior";
            } else if (name == "rough2" || name == "roughness_2") {
                info.is_rough_ior = false;
                info.rough_path2 = folder_prefix + element->Attribute("value");
            } else if (name == "normal") {
                info.normal_path = folder_prefix + element->Attribute("value");
            } else {
                std::cerr << "Unsupported texture type '" << name << "'\n";
                throw std::runtime_error("Unexpected texture type.");
            }
            element = element->NextSiblingElement("string");
        }
        texs.emplace(id, std::move(info));
        tex_elem = tex_elem->NextSiblingElement("texture");
    }
}

const std::array<std::string, NumRendererType> RENDER_TYPE_STR = {
    "MegaKernel-PT", 
    "Wavefront-PT", 
    "Megakernel-LT", 
    "Voxel-SDF-PT", 
    "Depth Tracer", 
    "BVH Cost Visualizer"
};

Scene::Scene(std::string path): num_bsdfs(0), num_emitters(0), num_objects(0), num_prims(0), envmap_id(0) {
    tinyxml2::XMLDocument doc;
    if (doc.LoadFile(path.c_str()) != tinyxml2::XML_SUCCESS) {
        std::cerr << "Failed to load file" << std::endl;
    }

    auto folder_prefix = get_folder_path(path);
    const tinyxml2::XMLElement  *scene_elem   = doc.FirstChildElement("scene"),
                                *acc_elem     = scene_elem->FirstChildElement("accelerator"), 
                                *bsdf_elem    = scene_elem->FirstChildElement("brdf"),
                                *shape_elem   = scene_elem->FirstChildElement("shape"),
                                *emitter_elem = scene_elem->FirstChildElement("emitter"),
                                *sensor_elem  = scene_elem->FirstChildElement("sensor"), 
                                *render_elem  = scene_elem->FirstChildElement("renderer"), 
                                *texture_elem = scene_elem->FirstChildElement("texture"), 
                                *bool_elem    = scene_elem->FirstChildElement("bool"), *ptr = nullptr;
    if (auto version_id = scene_elem->Attribute("version")) {
        if(std::strcmp(version_id, SCENE_VERSION) != 0) {
            std::cerr << "[SCENE] Version required: '" << SCENE_VERSION << "', got '" << version_id << "'. Abort.\n";
            exit(0);
        }
    }

    std::unordered_map<std::string, int> bsdf_map, emitter_map, emitter_obj_map;
    std::vector<std::string> emitter_names;
    emitter_names.reserve(9);
    emitter_map.reserve(9);
    bsdf_map.reserve(48);


    // ------------------------- (0) parse the renderer -------------------------
    std::string render_type = render_elem != nullptr ? render_elem->Attribute("type") : "pt";
    if      (render_type == "pt")    rdr_type = RendererType::MegaKernelPT;
    else if (render_type == "wfpt")  rdr_type = RendererType::WavefrontPT;
    else if (render_type == "lt")    rdr_type = RendererType::MegeKernelLT;
    else if (render_type == "sdf")   rdr_type = RendererType::VoxelSDFPT;
    else if (render_type == "depth") rdr_type = RendererType::DepthTracing;
    else if (render_type == "bvh-cost") rdr_type = RendererType::BVHCostViz;
    else                                rdr_type = RendererType::MegaKernelPT;
    
    // ------------------------- (1) parse all the textures and BSDF -------------------------
    
    std::unordered_map<std::string, TextureInfo> tex_map;
    parseTexture(texture_elem, tex_map, folder_prefix);

    ptr = bsdf_elem;
    for (; ptr != nullptr; ++ num_bsdfs)
        ptr = ptr->NextSiblingElement("brdf");
    if (num_bsdfs > MAX_ALLOWED_BSDF) {
        std::cerr << "Number of materials more than allowed. Max: " << MAX_ALLOWED_BSDF << std::endl;
        throw std::runtime_error("Too many BSDF defined.");
    }
    CUDA_CHECK_RETURN(hipMalloc(&bsdfs, sizeof(BSDF*) * num_bsdfs));

    textures.init(num_bsdfs);
    for (int i = 0; i < num_bsdfs; i++) {
        parseBSDF(bsdf_elem, tex_map, bsdf_map, bsdf_infos, host_tex_4d, host_tex_2d, textures, bsdfs, i);
        bsdf_elem = bsdf_elem->NextSiblingElement("brdf");
    }
    textures.to_gpu();
    CUDA_CHECK_RETURN(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_textures), &textures, sizeof(Textures), 0, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // ------------------------- (2) parse emitter names -------------------------
    parseEmitterNames(emitter_elem, emitter_map);

    // ------------------------- (3) parse all objects -------------------------
    ptr = shape_elem;
    for (; ptr != nullptr; ++ num_objects)
        ptr = ptr->NextSiblingElement("shape");
    objects.reserve(num_objects);

    std::vector<bool> sphere_objs(num_objects, false);

    for (int i = 0; i < 3; i++) {
        verts_list[i].reserve(32);
        norms_list[i].reserve(32);
        uvs_list[i].reserve(32);
    }

    int prim_offset = 0;
    for (int i = 0; i < num_objects; i++) {
        std::string type = shape_elem->Attribute("type");
        if (type == "obj")
            parseObjShape(shape_elem, bsdf_map, emitter_map, bsdf_infos, emitter_obj_map, 
                    objects, verts_list, norms_list, uvs_list, prim_offset, folder_prefix, i);
        else if (type == "sphere")
            parseSphereShape(shape_elem, bsdf_map, emitter_map, bsdf_infos, emitter_obj_map, 
                    objects, verts_list, norms_list, uvs_list, prim_offset, folder_prefix, i);
        sphere_objs[i] = type == "sphere";
        shape_elem = shape_elem->NextSiblingElement("shape");
    }
    num_prims = prim_offset;
    if (num_prims > MAX_PRIMITIVE_NUM) {
        // MAX_PRIMITIVE_NUM is the upper bound. 2^25 - 1, if num_prims exceeds this bound
        // For CompactNode, it is possible that the node offset will be out-of-range
        std::cerr << "[Error] Too many primitives: " << num_prims << " (maximum allowed: " << MAX_PRIMITIVE_NUM << ")\n";
        throw std::runtime_error("Too many primitives.");
    }

    //  ------------------------- (4) parse all emitters --------------------------
    ptr = emitter_elem;
    for (; ptr != nullptr; ++ num_emitters)
        ptr = ptr->NextSiblingElement("emitter");
    CUDA_CHECK_RETURN(hipMalloc(&emitters, sizeof(Emitter*) * (num_emitters + 1)));
    create_abstract_source<<<1, 1>>>(emitters[0]);
    for (int i = 1; i <= num_emitters; i++) {
        parseEmitter(emitter_elem, emitter_obj_map, tex_map, emitter_props, emitter_names, host_tex_4d, emitters, envmap_id, i);
        emitter_elem = emitter_elem->NextSiblingElement("emitter");
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // ------------------------- (5) parse camera & scene config -------------------------
    CUDA_CHECK_RETURN(hipHostMalloc(&cam, sizeof(DeviceCamera)));
    *cam = DeviceCamera::from_xml(sensor_elem);
    config = RenderingConfig::from_xml(acc_elem, render_elem, sensor_elem);

    // ------------------------- (6) initialize shapes -------------------------
    sphere_flags.resize(num_prims);
    prim_offset = 0;
    for (int obj_id = 0; obj_id < num_objects; obj_id ++) {
        prim_offset += objects[obj_id].prim_num;
        bool is_sphere = sphere_objs[obj_id];
        for (int i = objects[obj_id].prim_offset; i < prim_offset; i++) {
            sphere_flags[i] = is_sphere;
        }
    }

    printf("[BVH] Linear SAH-BVH is being built...\n");
    Vec3 world_min(AABB_INVALID_DIST), world_max(-AABB_INVALID_DIST);
    for (const auto& obj: objects) {
        obj.export_bound(world_min, world_max);
    }
    auto tp = std::chrono::system_clock::now();
    std::vector<int> prim_idxs;     // won't need this if BVH is built
    bvh_build(
        verts_list[0], verts_list[1], verts_list[2], 
        objects, sphere_objs, world_min, world_max, 
        obj_idxs, prim_idxs, nodes, 
        cache_nodes, config.cache_level, 
        config.max_node_num, config.bvh_overlap_w
    );
    auto dur = std::chrono::system_clock::now() - tp;
    auto count = std::chrono::duration_cast<std::chrono::microseconds>(dur).count();
    auto elapsed = static_cast<double>(count) / 1e3;
    printf("[BVH] BVH completed within %.3lf ms\n", elapsed);
    // The nodes.size is actually twice the number of nodes
    // since Each BVH node will be separated to two float4, nodes will store two float4 for each node
    printf("[BVH] Total nodes: %llu, leaves: %llu\n", nodes.size(), prim_idxs.size());

    tp = std::chrono::system_clock::now();
    std::array<Vec3Arr, 3> reorder_verts, reorder_norms;
    std::array<Vec2Arr, 3> reorder_uvs;
    std::vector<bool> reorder_sph_flags(num_prims);

    for (int i = 0; i < 3; i++) {
        Vec3Arr &reorder_vs = reorder_verts[i],
                &reorder_ns = reorder_norms[i];
        Vec2Arr &reorder_uv = reorder_uvs[i];
        const Vec3Arr &origin_vs = verts_list[i],
                        &origin_ns = norms_list[i];
        const Vec2Arr &origin_uv = uvs_list[i];
        reorder_vs.resize(num_prims);
        reorder_ns.resize(num_prims);
        reorder_uv.resize(num_prims);

        for (int j = 0; j < num_prims; j++) {
            int index = prim_idxs[j];
            reorder_vs[j] = origin_vs[index];
            reorder_ns[j] = origin_ns[index];
            reorder_uv[j] = origin_uv[index];
        }
    }

    // build an emitter primitive index map for emitter sampling
    // before the reordering logic, the emitter primitives are gauranteed
    // to be stored continuously, so we don't need an extra index map

    // if we don't reorder the primitives, then we need to store the primitive index
    // for the leaf node, and the access for the leaf node primitives won't be continuous
    std::vector<std::vector<int>> eprim_idxs(num_emitters);
    for (int i = 0; i < num_prims; i++) {
        int index = prim_idxs[i], obj_idx = obj_idxs[i];
        obj_idx = obj_idx < 0 ? -obj_idx - 1 : obj_idx;
        const auto& object = objects[obj_idx];
        reorder_sph_flags[i] = sphere_flags[index];
        if (object.is_emitter()) {
            int emitter_idx = object.emitter_id - 1;
            eprim_idxs[emitter_idx].push_back(i);
        }
    }
    // The following code does the following job:
    // BVH op will 'shuffle' the primitive order (sort of)
    // So, the emitter object might not have continuous
    // primitives stored in the memory. In order to uniformly sample
    // all the primitives on a given emitter, we should store the linearized
    // indices to the primitives, so the following code (1) linearize
    // the indices and (2) recalculate the object.prim_offset, while
    // the object.prim_cnt stays unchanged
    std::vector<int> e_prim_offsets;
    e_prim_offsets.push_back(0);
    for (const auto& eprim_idx: eprim_idxs) {
        e_prim_offsets.push_back(eprim_idx.size());
        for (int index: eprim_idx) 
            emitter_prims.push_back(index);
    }
    std::partial_sum(e_prim_offsets.begin(), e_prim_offsets.end(), e_prim_offsets.begin());
    for (ObjInfo& obj: objects) {
        if (!obj.is_emitter()) continue;
        obj.prim_offset = e_prim_offsets[obj.emitter_id - 1];
    }

    uvs_list     = std::move(reorder_uvs);
    verts_list   = std::move(reorder_verts);
    norms_list   = std::move(reorder_norms);
    sphere_flags = std::move(reorder_sph_flags);
    dur = std::chrono::system_clock::now() - tp;
    count = std::chrono::duration_cast<std::chrono::microseconds>(dur).count();
    elapsed = static_cast<double>(count) / 1e3;
    printf("[BVH] Vertex data reordering completed within %.3lf ms\n", elapsed);
}

Scene::~Scene() {
    destroy_gpu_alloc<<<1, num_bsdfs>>>(bsdfs);
    destroy_gpu_alloc<<<1, num_emitters + 1>>>(emitters);

    CUDA_CHECK_RETURN(hipFree(bsdfs));
    CUDA_CHECK_RETURN(hipFree(emitters));
    CUDA_CHECK_RETURN(hipHostFree(cam));
    for (auto& tex: host_tex_4d) tex.destroy();
    for (auto& tex: host_tex_2d) tex.destroy();
    textures.destroy();
}

CPT_KERNEL static void vec2_to_packed_half_kernel(const Vec2* src1, const Vec2* src2, const Vec2* src3, PackedHalf2* dst, size_t count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < count; i += blockDim.x * gridDim.x) {
        dst[i] = PackedHalf2(src1[i], src2[i], src3[i]);
    }
}

void Scene::update_emitters() {
    for (int index = 1; index <= num_emitters; index++) {
        Vec4 color = emitter_props[index - 1].second;
        if (color.x() < 0) {
            call_setter<<<1, 1>>>(emitters[index], color.y(), color.z() * DEG2RAD, color.w() * DEG2RAD);
        } else {
            set_emission<<<1, 1>>>(emitters[index], color.xyz(), color.w());
        }
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

void Scene::update_materials() {
    for (size_t i = 0; i < bsdf_infos.size(); i++) {
        auto& bsdf_info = bsdf_infos[i];
        if (bsdf_info.bsdf_changed) {
            bsdf_info.bsdf_value_clamping();
            bsdf_info.create_on_gpu(bsdfs[i]);
        } else {
            bsdf_info.copy_to_gpu(bsdfs[i]);
        }
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

template <typename T>
static void free_resource(std::vector<T>& vec) {
    vec.clear();
    vec.shrink_to_fit();
}

void Scene::free_resources() {
    for (int i = 0; i < 3; i++) {
        free_resource(verts_list[i]);
        free_resource(norms_list[i]);
        free_resource(uvs_list[i]);
    }
    free_resource(objects);
    free_resource(sphere_flags);
    free_resource(obj_idxs);
    free_resource(nodes);
    free_resource(cache_nodes);
    free_resource(emitter_prims);
}

void Scene::export_prims(PrecomputedArray& verts, NormalArray& norms, ConstBuffer<PackedHalf2>& uvs) const {
    verts.from_vectors(verts_list[0], verts_list[1], verts_list[2], &sphere_flags);
    norms.from_vectors(norms_list[0], norms_list[1], norms_list[2]);
    SoA3<Vec2> uvs_float(num_prims);
    uvs_float.from_vectors(uvs_list[0], uvs_list[1], uvs_list[2]);

    constexpr size_t block_size = 256;
    int num_blocks = (num_prims + block_size - 1) / block_size; // 计算所需 block 数
    vec2_to_packed_half_kernel<<<num_blocks, block_size>>>(&uvs_float.x(0), &uvs_float.y(0), &uvs_float.z(0), uvs.data(), num_prims);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    uvs_float.destroy();
}

void Scene::print() const noexcept {
    std::cout << " Rendering Settings:\n";
    std::cout << "\tRenderer type: " << RENDER_TYPE_STR[rdr_type] << std::endl;
    std::cout << "\t\tConfig: max depth:\t" << config.md.max_depth << std::endl;
    std::cout << "\t\tConfig: max diffuse:\t" << config.md.max_diffuse << std::endl;
    std::cout << "\t\tConfig: max specular:\t" << config.md.max_specular << std::endl;
    std::cout << "\t\tConfig: max transmit:\t" << config.md.max_tranmit << std::endl;
    std::cout << "\t\tConfig: Spec Cons:\t" << config.spec_constraint << std::endl;
    std::cout << "\t\tConfig: Bidirectional:\t" << config.bidirectional << std::endl;
    std::cout << "\t\tConfig: Caustics Scale:\t" << config.caustic_scaling << std::endl;
    std::cout << "\t\tConfig: SPP:\t\t" << config.spp << std::endl;
    std::cout << std::endl;

    std::cout << "\tAccelerator type: BVH" << std::endl;
    std::cout << "\t\tSAH-BVH Cache Level: \t" << config.cache_level << std::endl;
    std::cout << "\t\tBVH Max Leaf Node: \t" << config.max_node_num << std::endl;
    std::cout << "\t\tBVH Overlap Weight: \t" << config.bvh_overlap_w << std::endl;
    std::cout << std::endl;

    std::cout << "\tScene statistics: " << std::endl;
    std::cout << "\t\tNumber of objects: \t" << num_objects << std::endl;
    std::cout << "\t\tNumber of primitives: \t" << num_prims << std::endl;
    std::cout << "\t\tNumber of emitters: \t" << num_emitters << std::endl;
    std::cout << "\t\tNumber of BSDFs: \t" << num_bsdfs << std::endl;
    std::cout << std::endl;
    std::cout << "\tCamera Film Configs: " << std::endl;
    std::cout << "\t\tConfig: width:\t\t" << config.width << std::endl;
    std::cout << "\t\tConfig: height:\t\t" << config.height << std::endl;
    std::cout << "\t\tConfig: Gamma corr:\t" << config.gamma_correction << std::endl;
    std::cout << std::endl;
}