#include "hip/hip_runtime.h"
// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Spatial BVH construction main logic
 * @date 2025.5.25
 */

#include "core/bvh_opt.cuh"
#include "core/bvh_spatial.cuh"
#include "core/spatial"
#include <numeric>

static constexpr int num_bins = 16;
static constexpr int no_div_threshold = 2;
static constexpr int sah_split_threshold = 8;
// A cluster with all the primitive centroid within a small range [less than
// 1e-3] is ill-posed. If there is more than 64 primitives, the primitives will
// be discarded
static constexpr float traverse_cost = 0.2f;
static constexpr int unordered_threshold = 512;

static float bvh_overlap_w = 1.f;
static int max_depth = 0;

SplitAxis SBVHNode::max_extent_axis(const std::vector<BVHInfo> &bvhs,
                                    float &min_r, float &interval) const {

    Vec3 min_ctr = Vec3(std::numeric_limits<float>::max()),
         max_ctr = Vec3(std::numeric_limits<float>::min());

    for (int bvh_id : prims) {
        Vec3 ctr = bvhs[bvh_id].centroid;
        min_ctr.minimized(ctr);
        max_ctr.maximized(ctr);
    }

    Vec3 diff = max_ctr - min_ctr;
    float max_diff = diff.x();
    min_r = min_ctr[0] - 1e-5;
    int split_axis = 0;
    if (diff.y() > max_diff) {
        max_diff = diff.y();
        split_axis = 1;
        min_r = min_ctr[1] - 1e-5;
    }
    if (diff.z() > max_diff) {
        max_diff = diff.z();
        split_axis = 2;
        min_r = min_ctr[2] - 1e-5;
    }
    if (diff.max_elem() < 1e-3) {
        return SplitAxis::AXIS_NONE;
    }
    interval = (max_diff + 2e-5f) / float(num_bins);
    return SplitAxis(split_axis);
}

inline int object_index_packing(int obj_med_idx, int obj_id, bool is_sphere) {
    // take the lower 20 bits and shift up 20bits
    int truncated = (obj_med_idx & 0x00000fff) << 20;
    return (static_cast<int>(is_sphere) << 31) + truncated +
           (obj_id & 0x000fffff);
}

// TODO(heqianyue): note that we currently don't support
// sphere primitive. Support it would be straightforward:
// overload the 'update' function for spheres
int recursive_sbvh_SAH(const std::vector<Vec3> &points1,
                       const std::vector<Vec3> &points2,
                       const std::vector<Vec3> &points3,
                       SBVHNode *const cur_node,
                       std::vector<BVHInfo> &bvh_infos, int depth = 0,
                       int max_prim_node = 16) {
    AABB fwd_bound(1e5f, -1e5f, 0, 0), bwd_bound(1e5f, -1e5f, 0, 0);
    int child_prim_cnt = 0; // this index is used for indexing variable `bins`
    const int prim_num = cur_node->prim_num();
    float min_cost = 5e9f, node_prim_cnt = float(prim_num);

    // Step 1: decide the axis that expands the maximum extent of space
    float min_range = 0, interval = 0;
    SplitAxis max_axis =
        cur_node->max_extent_axis(bvh_infos, min_range, interval);

    if (max_axis != SplitAxis::AXIS_NONE &&
        prim_num > sah_split_threshold) { // SAH
        // Step 2: binning the space
        std::array<AxisBins, num_bins> idx_bins;
        for (int bvh_id : cur_node->prims) {
            int index = std::min(
                (int)floorf((bvh_infos[bvh_id].centroid[max_axis] - min_range) /
                            interval),
                num_bins - 1);
            idx_bins[index].push(bvh_infos[bvh_id]);
        }

        // Step 3: forward-backward linear sweep for heuristic calculation
        std::array<int, num_bins> prim_cnts;
        std::array<float, num_bins> fwd_areas, bwd_areas;
        std::vector<AABB> fwd_aabbs,
            bwd_aabbs; // to calculate AABB intersection
        fwd_aabbs.reserve(num_bins);
        bwd_aabbs.reserve(num_bins);
        prim_cnts.fill(0);
        fwd_areas.fill(0);
        bwd_areas.fill(0);
        for (int i = 0; i < num_bins; i++) {
            fwd_bound += idx_bins[i].bound;
            prim_cnts[i] = idx_bins[i].prim_cnt;
            fwd_areas[i] = fwd_bound.area();
            fwd_aabbs.push_back(fwd_bound);
            if (i > 0) {
                bwd_bound += idx_bins[num_bins - i].bound;
                bwd_areas[num_bins - 1 - i] = bwd_bound.area();
                bwd_aabbs.push_back(bwd_bound);
            }
        }
        cur_node->bound.mini = fwd_bound.mini;
        cur_node->bound.maxi = fwd_bound.maxi;
        float node_inv_area = 1. / cur_node->bound.area();
        std::partial_sum(prim_cnts.begin(), prim_cnts.end(), prim_cnts.begin());

        // Step 4: use the calculated area to computed the segment boundary
        int seg_bin_idx = 0;
        for (int i = 0; i < num_bins - 1; i++) {
            float intrsct_a = fwd_aabbs[i].intersection_area(bwd_aabbs.back());
            float cost =
                traverse_cost +
                node_inv_area *
                    (intrsct_a * std::max(bvh_overlap_w - 0.5f, 0.f) *
                         node_prim_cnt +
                     float(prim_cnts[i]) * fwd_areas[i] +
                     (node_prim_cnt - float(prim_cnts[i])) * bwd_areas[i]);
            if (cost < min_cost) {
                min_cost = cost;
                seg_bin_idx = i;
            }
            bwd_aabbs.pop_back();
        }

        int sbvh_seg_idx = -1;
        if (false /*Some unknown criteria I didn't come up with yet*/) {
            // if the crieria are met, we calculate the SBVH split cost
            SpatialSplitter<num_bins> ssp(
                min_range, min_range + interval * static_cast<float>(num_bins),
                max_axis);

            ssp.update_bins(points1, points2, points3, cur_node);

            float sbvh_cost =
                ssp.eval_spatial_split(cur_node, sbvh_seg_idx, traverse_cost);
            if (sbvh_cost < min_cost) { // Spatial split should be applied
                // ssp.apply_spatial_split(cur_node, );
            }
        }
        // Step 5: reordering the BVH info in the vector to make the segment
        // contiguous (partition around pivot)
        if (min_cost < node_prim_cnt || prim_num > max_prim_node) {
            std::partition(
                bvh_infos.begin() + base, bvh_infos.begin() + max_pos,
                [pivot = min_range + interval * float(seg_bin_idx + 1),
                 dim = max_axis](const BVHInfo &bvh) {
                    return bvh.centroid[dim] < pivot;
                });
            child_prim_cnt = prim_cnts[seg_bin_idx];
        }

        fwd_bound.clear();
        bwd_bound.clear();
        for (int i = 0; i <= seg_bin_idx; i++) // calculate child node bound
            fwd_bound += idx_bins[i].bound;
        for (int i = seg_bin_idx + 1; i < num_bins; i++)
            bwd_bound += idx_bins[i].bound;
    } else { // equal primitive number split (two nodes have identical
             // primitives)
        int seg_idx = (base + max_pos) >> 1;
        // Step 5: reordering the BVH info in the vector to make the segment
        // contiguous (keep around half of the bvh in lchild)
        if (max_axis != SplitAxis::AXIS_NONE) {
            std::nth_element(
                bvh_infos.begin() + base, bvh_infos.begin() + seg_idx,
                bvh_infos.begin() + max_pos,
                [dim = max_axis](const BVHInfo &bvh1, const BVHInfo &bvh2) {
                    return bvh1.centroid[dim] < bvh2.centroid[dim];
                });
        }
        for (int i = base; i < seg_idx; i++) // calculate child node bound
            fwd_bound += bvh_infos[i].bound;
        for (int i = seg_idx; i < max_pos; i++)
            bwd_bound += bvh_infos[i].bound;
        cur_node->bound += fwd_bound;
        cur_node->bound += bwd_bound;
        child_prim_cnt = seg_idx - base; // bvh[seg_idx] will be in rchild
        float intrsct_a = fwd_bound.intersection_area(bwd_bound);
        float split_cost =
            traverse_cost +
            (1.f / cur_node->bound.area()) *
                (intrsct_a * std::max(bvh_overlap_w - 0.5f, 0.f) *
                     node_prim_cnt +
                 fwd_bound.area() * float(child_prim_cnt) +
                 bwd_bound.area() * (node_prim_cnt - float(child_prim_cnt)));
        if (split_cost >= node_prim_cnt && prim_num < max_prim_node)
            child_prim_cnt = 0;
    }

    if (child_prim_cnt >
        0) { // cost of splitting is less than making this node a leaf node
        // Step 5: split the node and initialize the children
        cur_node->lchild = new BVHNode(base, child_prim_cnt);
        cur_node->rchild =
            new BVHNode(base + child_prim_cnt, prim_num - child_prim_cnt);

        cur_node->lchild->bound.mini = fwd_bound.mini;
        cur_node->lchild->bound.maxi = fwd_bound.maxi;
        cur_node->rchild->bound.mini = bwd_bound.mini;
        cur_node->rchild->bound.maxi = bwd_bound.maxi;
        cur_node->axis = max_axis;
        // Step 7: start recursive splitting for the children
        int node_num = 1;
        if (cur_node->lchild->prim_num() > no_div_threshold)
            node_num += recursive_bvh_SAH(cur_node->lchild, bvh_infos,
                                          depth + 1, max_prim_node);
        else {
            max_depth = std::max(depth + 1, max_depth);
            node_num++;
        }
        if (cur_node->rchild->prim_num() > no_div_threshold)
            node_num += recursive_bvh_SAH(cur_node->rchild, bvh_infos,
                                          depth + 1, max_prim_node);
        else {
            max_depth = std::max(depth + 1, max_depth);
            node_num++;
        }
        return node_num;
    } else {
        // This is a leaf node, yet this is the only way that a leaf node
        // contains more than one primitive
        cur_node->axis = AXIS_NONE;
        max_depth = std::max(depth, max_depth);
        return 1;
    }
}

static BVHNode *bvh_root_start(const Vec3 &world_min, const Vec3 &world_max,
                               int &node_num, std::vector<BVHInfo> &bvh_infos,
                               int max_prim_node = 16) {
    // Build BVH tree root node and start recursive tree construction
    printf("[BVH] World min: ");
    print_vec3(world_min);
    printf("[BVH] World max: ");
    print_vec3(world_max);
    BVHNode *root_node = new BVHNode(0, bvh_infos.size());
    root_node->bound.mini = world_min;
    root_node->bound.maxi = world_max;
    node_num = recursive_bvh_SAH(root_node, bvh_infos, max_prim_node);
    return root_node;
}

// This is the final function call for `bvh_build`
static int recursive_linearize(BVHNode *cur_node, std::vector<float4> &nodes,
                               std::vector<CompactNode> &cache_nodes,
                               const int depth = 0,
                               const int cache_max_depth = 4) {
    // BVH tree should be linearized to better traverse and fit in the system
    // memory The linearized BVH tree should contain: bound, base, prim_cnt,
    // rchild_offset, total_offset (to skip the entire node) Note that if
    // rchild_offset is -1, then the node is leaf. Leaf node points to primitive
    // array which is already sorted during BVH construction, containing
    // primitive_id and obj_id for true intersection Note that lin_nodes has
    // been reserved
    size_t current_size = nodes.size() >> 1,
           current_cached = cache_nodes.size();
    float4 node_f, node_b;
    cur_node->get_float4(node_f, node_b);
    nodes.push_back(node_f);
    nodes.push_back(node_b);
    reinterpret_cast<uint32_t &>(node_f.w) =
        1; // always assume leaf node (offset = 1)
    reinterpret_cast<uint32_t &>(node_b.w) = current_size;
    if (depth < cache_max_depth) {
        // LinearNode (cached):
        // (float3) aabb.min
        // (int)    jump offset to next cached node
        // (float3) aabb.max
        // (int)    index to the global memory node (if -1, means it it not a
        // leave node, we should continue)
        cache_nodes.emplace_back(node_f, node_b);
    }
    /**
     * @note
     * Clarify on how do we store BVH range and node offsets:
     * - for non-leaf nodes, since beg_idx and end_idx will not be used, we only
     * need node_offset SO node_offset is stored as the `NEGATIVE` value, so if
     * we encounter a negative float4.w, we know that the current node is
     * non-leaf
     * - for leaf nodes, we don't modify the float4.w
     */
    if (cur_node->lchild != nullptr) {
        // non-leaf node
        int lnodes = recursive_linearize(cur_node->lchild, nodes, cache_nodes,
                                         depth + 1, cache_max_depth);
        lnodes += recursive_linearize(cur_node->rchild, nodes, cache_nodes,
                                      depth + 1, cache_max_depth);
        INT_REF_CAST(nodes[2 * current_size + 1].w) = -(lnodes + 1);
        if (depth < cache_max_depth) {
            // store the jump offset to the next cached node (for non-leaf node)
            cache_nodes[current_cached].set_low_8bits(cache_nodes.size() -
                                                      current_cached);
        }
        return lnodes + 1; // include the cur_node
    } else {
        // leaf node has negative offset
        return 1;
    }
}

// Try to use two threads to build the BVH
void bvh_build(const std::vector<Vec3> &points1,
               const std::vector<Vec3> &points2,
               const std::vector<Vec3> &points3,
               const std::vector<ObjInfo> &objects,
               const std::vector<int> &obj_med_idxs,
               const std::vector<bool> &sphere_flags, const Vec3 &world_min,
               const Vec3 &world_max, std::vector<int> &obj_idxs,
               std::vector<int> &prim_idxs, std::vector<float4> &nodes,
               std::vector<CompactNode> &cache_nodes, int &cache_max_level,
               const int max_prim_node, const float overlap_w) {
    bvh_overlap_w = overlap_w;
    std::vector<PrimMappingInfo> idx_prs;
    std::vector<BVHInfo> bvh_infos;
    int node_num = 0, num_prims_all = points1.size();
    index_input(objects, sphere_flags, idx_prs, num_prims_all);
    create_bvh_info(points1, points2, points3, idx_prs, obj_med_idxs,
                    bvh_infos);
    BVHNode *root_node = bvh_root_start(world_min, world_max, node_num,
                                        bvh_infos, max_prim_node);
    float total_cost = calculate_cost(root_node, traverse_cost);

    printf("[BVH] BVH tree max depth: %d\n", max_depth);
    printf("[BVH] Traversed BVH SAH cost: %.7f, AVG: %.7f\n", total_cost,
           total_cost / static_cast<float>(bvh_infos.size()));
    cache_max_level = std::min(std::max(max_depth - 1, 0), cache_max_level);
    nodes.reserve(node_num << 1);
    cache_nodes.reserve(1 << cache_max_level);
    recursive_linearize(root_node, nodes, cache_nodes, 0, cache_max_level);
    printf("[BVH] Number of nodes to cache: %llu (%d)\n", cache_nodes.size(),
           cache_max_level);

    // FIXME: MASK ALPHA, change obj_idxs
    obj_idxs.reserve(bvh_infos.size());
    prim_idxs.reserve(bvh_infos.size());
    for (BVHInfo &bvh : bvh_infos) {
        obj_idxs.emplace_back(bvh.bound.__bytes1);
        prim_idxs.emplace_back(bvh.bound.__bytes2);
    }
    delete root_node;
}
