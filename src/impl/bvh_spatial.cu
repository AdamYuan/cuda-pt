#include "hip/hip_runtime.h"
// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Spatial BVH construction main logic
 * @date 2025.5.25
 */

#include "core/bvh_opt.cuh"
#include "core/bvh_spatial.cuh"
#include <numeric>
#include <unordered_set>

static constexpr int num_bins = 16;
static constexpr int no_div_threshold = 2;
static constexpr int sah_split_threshold = 8;
// A cluster with all the primitive centroid within a small range [less than
// 1e-3] is ill-posed. If there is more than 64 primitives, the primitives will
// be discarded
static constexpr float traverse_cost = 0.2f;
static constexpr bool SSP_DEBUG = false;
static constexpr int max_allowed_depth = 96;
static int max_depth = 0;

SplitAxis SBVHNode::max_extent_axis(const std::vector<BVHInfo> &bvhs,
                                    float &min_r, float &interval) const {

    Vec3 min_ctr = Vec3(std::numeric_limits<float>::max()),
         max_ctr = Vec3(-std::numeric_limits<float>::max());

    for (int bvh_id : prims) {
        Vec3 ctr = bvhs[bvh_id].centroid;
        min_ctr.minimized(ctr);
        max_ctr.maximized(ctr);
    }

    Vec3 diff = max_ctr - min_ctr;
    float max_diff = diff.x();
    min_r = min_ctr[0] - 1e-5;
    int split_axis = 0;
    if (diff.y() > max_diff) {
        max_diff = diff.y();
        split_axis = 1;
        min_r = min_ctr[1] - 1e-5;
    }
    if (diff.z() > max_diff) {
        max_diff = diff.z();
        split_axis = 2;
        min_r = min_ctr[2] - 1e-5;
    }
    if (diff.max_elem() < 1e-3) {
        return SplitAxis::AXIS_NONE;
    }
    interval = (max_diff + 2e-5f) / float(num_bins);
    return SplitAxis(split_axis);
}

template <int N>
void SpatialSplitter<N>::update_triangle(Vec3 v1, Vec3 v2, Vec3 v3,
                                         int prim_id) {
    // FIXME: we must confine the triangle inside of the box (even if
    // the triangle has some part that is outside of the AABB, that
    // part must not be considered in any way. Note that the current
    // implementation can be replaced by 3-line intersection algorithm, which is
    // also cheap to calculate by simple line-drawing.

    // 1. sort the points according to the position on the split axis
    // we won't have degenerate triangles here.
    float p1_v = v1[axis], p2_v = v2[axis], p3_v = v3[axis];
    Vec3 p1, p2, p3;
    if (p1_v > p2_v) {
        std::swap(v1, v2);
        std::swap(p1_v, p2_v); // make sure p1_v <= p2_v
    }
    // now v1[axis] <= v2[axis] always holds,
    // therefore, p1 can never be v2
    if (p3_v < p1_v) {
        p1 = v3;
        p2 = v1;
        p3 = v2;
    } else {
        p1 = v1;
        if (p2_v < p3_v) {
            p2 = v2;
            p3 = v3;
        } else {
            p2 = v3;
            p3 = v2;
        }
    }
    // After sorting, p1, p2, p3 should have increasing split axis coord
    // convert the abs position to direction and normalize
    Vec3 dir1 = p2 - p1;
    Vec3 dir2 = p3 - p1;
    dir1 *= 1.f / dir1[axis];
    dir2 *= 1.f / dir2[axis];

    // 2. get bin ID of p1, p2 and p3 and update the ID record
    int v1_id = get_bin_id(p1);
    int v2_id = get_bin_id(p2);
    int v3_id = get_bin_id(p3);
    enter_tris[v1_id].push_back(prim_id);
    exit_tris[v3_id].push_back(prim_id);

    float d2bin_start = s_pos + interval * static_cast<float>(v1_id) - p1[axis];
    Vec3 end_p1 = p1 + d2bin_start * dir1, end_p2 = p1 + d2bin_start * dir2;
    for (int id = v1_id; id <= v3_id; id++) {
        AABB &aabb = bounds[id];

        if (id != v1_id) {
            aabb.extend(end_p1);
            aabb.extend(end_p2);
        } else {
            aabb.extend(p1);
        }

        if (id == v2_id) {
            aabb.extend(p2);
            // reset the direction and normalize
            dir1 = p3 - p2;
            dir1 *= 1.f / dir1[axis];
            // reset end point 1
            end_p1 = p2 + dir1 * (s_pos + interval * static_cast<float>(v2_id) -
                                  p2[axis]);
        }

        if (id != v3_id) {
            end_p1 += interval * dir1;
            end_p2 += interval * dir2;
            aabb.extend(end_p1);
            aabb.extend(end_p2);
        } else {
            aabb.extend(p3);
        }
    }
}

template <int N>
void SpatialSplitter<N>::update_bins(const std::vector<Vec3> &points1,
                                     const std::vector<Vec3> &points2,
                                     const std::vector<Vec3> &points3,
                                     /* possibly, add sphere flag later */
                                     const SBVHNode *const cur_node) {
    for (int prim_id : cur_node->prims) {
        update_triangle(points1[prim_id], points2[prim_id], points3[prim_id],
                        prim_id);
    }
}

template <int N>
float SpatialSplitter<N>::eval_spatial_split(const SBVHNode *const cur_node,
                                             int &seg_bin_idx,
                                             float traverse_cost) {
    float min_cost = 5e9f, node_prim_cnt = float(cur_node->prim_num());

    std::array<float, N> fwd_areas, bwd_areas;
    prim_cnts.fill(0);
    fwd_areas.fill(0);
    bwd_areas.fill(0);

    AABB fwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0),
        bwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0);
    for (int i = 0; i < N; i++) {
        fwd_bound += bounds[i];
        prim_cnts[i] = enter_tris[i].size();
        fwd_areas[i] = fwd_bound.area();
        if (i > 0) {
            prim_cnts[i] += prim_cnts[i - 1];
            bwd_bound += bounds[N - i];
            bwd_areas[N - 1 - i] = bwd_bound.area();
        }
    }
    float node_inv_area = 1. / cur_node->bound.area();

    for (int i = 0; i < N - 1; i++) {
        float cost = traverse_cost +
                     node_inv_area *
                         (float(prim_cnts[i]) * fwd_areas[i] +
                          (node_prim_cnt - float(prim_cnts[i])) * bwd_areas[i]);
        if (cost < min_cost) {
            min_cost = cost;
            seg_bin_idx = i;
        }
    }
    return min_cost;
}

template <int N>
std::pair<AABB, AABB> SpatialSplitter<N>::apply_spatial_split(
    const SBVHNode *const cur_node, std::vector<int> &left_prims,
    std::vector<int> &right_prims, int seg_bin_idx) {
    const int prim_num = cur_node->prim_num();
    left_prims.reserve(prim_cnts[seg_bin_idx]);
    right_prims.reserve(prim_num / 2);
    std::unordered_set<int> exit_from_left;
    for (int i = 0; i <= seg_bin_idx; i++) {
        left_prims.insert(left_prims.begin(), enter_tris[i].begin(),
                          enter_tris[i].end());
        for (int v : exit_tris[i]) {
            exit_from_left.emplace(v);
        }
    }
    for (int prim_id : cur_node->prims) {
        if (exit_from_left.count(prim_id))
            continue;
        right_prims.push_back(prim_id);
    }

    if constexpr (SSP_DEBUG) {
        if (left_prims.empty() || right_prims.empty()) {
            std::cerr << "Spatial split results in empty child nodes: "
                      << left_prims.size() << ", " << right_prims.size()
                      << std::endl;
            throw std::runtime_error("Spatial split failed.");
        }
    }

    AABB fwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0),
        bwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0);
    fwd_bound.clear();
    bwd_bound.clear();
    for (int i = 0; i <= seg_bin_idx; i++) // calculate child node bound
        fwd_bound += bounds[i];
    for (int i = seg_bin_idx + 1; i < N; i++)
        bwd_bound += bounds[i];
    return std::make_pair(fwd_bound, bwd_bound);
}

bool spatial_split_criteria(float root_area, float cur_area, float intrs_area,
                            int depth) {
    // SS can only be applied when depth >= the following
    static constexpr int spatial_split_depth = 1;
    // SS can be applied if local overlap >= the following
    static constexpr float local_overlap_factor = 0.5;
    // SS can be applied if overlap relative to root >= the following. This
    // factor is in fact mentioned in the original paper.
    static constexpr float root_overlap_factor = 1e-5f;

    return (depth >= spatial_split_depth) &&
           ((intrs_area > cur_area * local_overlap_factor) ||
            (intrs_area > root_overlap_factor * root_area));
}

// TODO(heqianyue): note that we currently don't support
// sphere primitive. Support it would be straightforward:
// overload the 'update' function for spheres
int recursive_sbvh_SAH(const std::vector<Vec3> &points1,
                       const std::vector<Vec3> &points2,
                       const std::vector<Vec3> &points3,
                       const std::vector<BVHInfo> &bvh_infos,
                       std::vector<int> &flattened_idxs,
                       SBVHNode *const cur_node, float root_area, int depth = 0,
                       int max_prim_node = 16) {
    auto process_leaf = [&]() {
        // leaf node processing function
        cur_node->axis = AXIS_NONE;
        cur_node->base() = static_cast<int>(flattened_idxs.size());
        cur_node->prim_num() = static_cast<int>(cur_node->prims.size());
        max_depth = std::max(depth, max_depth);
        for (int prim_id : cur_node->prims) {
            flattened_idxs.push_back(prim_id);
        }
        // TODO(heqianyue): check whether the leaf node has valid bound
        return 1;
    };

    if (cur_node->size() <= no_div_threshold) {
        return process_leaf();
    }
    AABB fwd_bound(1e5f, -1e5f, 0, 0), bwd_bound(1e5f, -1e5f, 0, 0);
    const int prim_num = cur_node->size();
    float min_cost = 5e9f, node_prim_cnt = float(prim_num);

    // Step 1: decide the axis that expands the maximum extent of space
    float min_range = 0, interval = 0;
    SplitAxis max_axis =
        cur_node->max_extent_axis(bvh_infos, min_range, interval);

    std::vector<int> lchild_idxs, rchild_idxs;

    // lchild_idxs.reserve(prim_num / 2);
    // rchild_idxs.reserve(prim_num / 2);

    if (true) { // SAH
        // Step 2: binning the space
        std::array<AxisBins, num_bins> idx_bins;
        for (int bvh_id : cur_node->prims) {
            int index = std::min(
                (int)floorf((bvh_infos[bvh_id].centroid[max_axis] - min_range) /
                            interval),
                num_bins - 1);
            idx_bins[index].push(bvh_infos[bvh_id]);
        }

        // Step 3: forward-backward linear sweep for heuristic calculation
        std::array<int, num_bins> prim_cnts;
        std::array<float, num_bins> fwd_areas, bwd_areas;

        prim_cnts.fill(0);
        fwd_areas.fill(0);
        bwd_areas.fill(0);
        for (int i = 0; i < num_bins; i++) {
            fwd_bound += idx_bins[i].bound;
            prim_cnts[i] = idx_bins[i].prim_cnt;
            fwd_areas[i] = fwd_bound.area();
            if (i > 0) {
                bwd_bound += idx_bins[num_bins - i].bound;
                bwd_areas[num_bins - 1 - i] = bwd_bound.area();
            }
        }
        cur_node->bound.mini = fwd_bound.mini;
        cur_node->bound.maxi = fwd_bound.maxi;
        float node_inv_area = 1. / cur_node->bound.area();
        std::partial_sum(prim_cnts.begin(), prim_cnts.end(), prim_cnts.begin());

        // Step 4: use the calculated area to computed the segment boundary, for
        // SBVH there is no need using spatial overlap penalty for BVH
        int seg_bin_idx = 0;
        for (int i = 0; i < num_bins - 1; i++) {
            float cost =
                traverse_cost +
                node_inv_area *
                    (float(prim_cnts[i]) * fwd_areas[i] +
                     (node_prim_cnt - float(prim_cnts[i])) * bwd_areas[i]);
            if (cost < min_cost) {
                min_cost = cost;
                seg_bin_idx = i;
            }
        }

        fwd_bound.clear();
        bwd_bound.clear();
        for (int i = 0; i <= seg_bin_idx; i++) // calculate child node bound
            fwd_bound += idx_bins[i].bound;
        for (int i = seg_bin_idx + 1; i < num_bins; i++)
            bwd_bound += idx_bins[i].bound;

        if (spatial_split_criteria(root_area, cur_node->bound.area(),
                                   fwd_bound.intersection_area(bwd_bound),
                                   depth)) {

            // TODO(heqianyue): there are still some optimization that can be
            // implemented. (1) Reference unsplitting. Since split one primitive
            // reference into two nodes when the reference introduces little
            // overlap, we can unsplit the reference.

            SpatialSplitter<num_bins> ssp(cur_node->bound);

            ssp.update_bins(points1, points2, points3, cur_node);

            int sbvh_seg_idx = 0;
            float sbvh_cost =
                ssp.eval_spatial_split(cur_node, sbvh_seg_idx, traverse_cost);
            if (sbvh_cost < min_cost &&
                (sbvh_cost < node_prim_cnt ||
                 prim_num > max_prim_node)) { // Spatial split should be applied
                min_cost = sbvh_cost;
                std::tie(fwd_bound, bwd_bound) = ssp.apply_spatial_split(
                    cur_node, lchild_idxs, rchild_idxs, sbvh_seg_idx);
            }
        }

        // 1. SBVH is not applied ; 2. when the cost of splitting is lower or 3.
        // when there are more primitives than allowed
        if (lchild_idxs.empty() && (min_cost < node_prim_cnt ||
                                    prim_num > max_prim_node)) { // object split
            // We cannot partition here, since partition will change the index
            // of the BVH
            float pivot = min_range + interval * float(seg_bin_idx + 1);
            for (int bvh_id : cur_node->prims) {
                const BVHInfo &bvh = bvh_infos[bvh_id];
                if (bvh.centroid[max_axis] < pivot) {
                    lchild_idxs.push_back(bvh_id);
                } else {
                    rchild_idxs.push_back(bvh_id);
                }
            }
        }
    } else { // equal primitive number split (two nodes have identical
             // primitives)
        std::vector<std::pair<float, int>> valued_indices;
        // valued_indices.reserve(cur_node->size());
        // printf("valued_indices reserve: %d\n", cur_node->size());
        for (int bvh_id : cur_node->prims) {
            valued_indices.emplace_back(bvh_infos[bvh_id].centroid[max_axis],
                                        bvh_id);
        }

        // Step 5: reordering the BVH info in the vector to make the segment
        // contiguous (keep around half of the bvh in lchild)
        int half_size = valued_indices.size() / 2;
        // printf("curnode: %d, valued index: %lu, node size: %lu, half size:
        // %lu\n", int(cur_node == nullptr), valued_indices.size(),
        // cur_node->size(), half_size);
        std::sort(
            valued_indices.begin(), valued_indices.end(),
            [](const auto &a, const auto &b) { return a.first < b.first; });

        for (int i = 0; i < half_size; i++) {
            int bvh_id = valued_indices[i].second;
            lchild_idxs.push_back(bvh_id);
            fwd_bound += bvh_infos[bvh_id].bound;
        }
        for (int i = half_size; i < valued_indices.size(); i++) {
            int bvh_id = valued_indices[i].second;
            rchild_idxs.push_back(bvh_id);
            bwd_bound += bvh_infos[bvh_id].bound;
        }
        cur_node->bound += fwd_bound;
        cur_node->bound += bwd_bound;
        float split_cost =
            traverse_cost +
            (1.f / cur_node->bound.area()) *
                (fwd_bound.area() * float(half_size) +
                 bwd_bound.area() * float(valued_indices.size() - half_size));
        if (split_cost >= node_prim_cnt && prim_num < max_prim_node)
            lchild_idxs.clear();
    }

    if (!lchild_idxs.empty() &&
        !rchild_idxs.empty()) { // cost of splitting is less than making this
                                // node a leaf node
        cur_node->release();    // release mem for non-leaf nodes
        cur_node->lchild =
            new SBVHNode(std::move(fwd_bound), std::move(lchild_idxs));
        cur_node->rchild =
            new SBVHNode(std::move(bwd_bound), std::move(rchild_idxs));
        cur_node->axis = max_axis;

        int node_num = 1;
        printf("recursive left, depth: %d, lchild (%lu): [", depth + 1,
               cur_node->lchild->prim_num());
        for (int v : cur_node->lchild->prims) {
            printf("%d, ", v);
        }
        printf("]\n");
        node_num += recursive_sbvh_SAH(points1, points2, points3, bvh_infos,
                                       flattened_idxs, cur_node->lchild,
                                       root_area, depth + 1, max_prim_node);

        node_num += recursive_sbvh_SAH(points1, points2, points3, bvh_infos,
                                       flattened_idxs, cur_node->rchild,
                                       root_area, depth + 1, max_prim_node);
        return node_num;
    } else {
        return process_leaf();
    }
}

static SBVHNode *sbvh_root_start(const std::vector<Vec3> &points1,
                                 const std::vector<Vec3> &points2,
                                 const std::vector<Vec3> &points3,
                                 const Vec3 &world_min, const Vec3 &world_max,
                                 std::vector<int> &flattened_idxs,
                                 std::vector<BVHInfo> &bvh_infos, int &node_num,
                                 int max_prim_node = 16) {
    // Build BVH tree root node and start recursive tree construction
    printf("[SBVH] World min: ");
    print_vec3(world_min);
    printf("[SBVH] World max: ");
    print_vec3(world_max);
    std::vector<int> all_prims(points1.size());
    std::iota(all_prims.begin(), all_prims.end(), 0);
    SBVHNode *root_node = new SBVHNode(
        AABB(world_min, world_max, 0, points1.size()), std::move(all_prims));
    node_num = recursive_sbvh_SAH(points1, points2, points3, bvh_infos,
                                  flattened_idxs, root_node,
                                  root_node->bound.area(), 0, max_prim_node);

    return root_node;
}

template <typename ContainerTy, size_t Dim = 3>
void remap_helper_func(const std::vector<int> &flattened_idxs,
                       ContainerTy &source) {
    static constexpr int n_threads = 4;
    const size_t num_new_prims = flattened_idxs.size();
    const size_t padded_size =
        (num_new_prims + n_threads - 1) / n_threads; // workload for each thread

    ContainerTy mapped_vals;
    if constexpr (Dim == 1) {
        mapped_vals.resize(num_new_prims);
    } else {
        for (int i = 0; i < 3; i++) {
            mapped_vals[i].resize(num_new_prims);
        }
    }
#pragma omp parallel for num_threads(n_threads)
    for (int tid = 0; tid < n_threads; tid++) {
        const size_t s_pos = tid * padded_size,
                     e_pos = std::min(s_pos + padded_size, num_new_prims);
        if constexpr (Dim == 1) {
            for (size_t i = s_pos; i < e_pos; i++) {
                int index = flattened_idxs[i];
                mapped_vals[i] = source[index];
            }
        } else {
#pragma unroll
            for (int dim = 0; dim < Dim; dim++) {
                const auto &old_vec = source[dim];
                auto &new_vec = mapped_vals[dim];
                for (size_t i = s_pos; i < e_pos; i++) {
                    int index = flattened_idxs[i];
                    new_vec[i] = old_vec[index];
                }
            }
        }
    }
    source = std::move(mapped_vals);
}

void SBVHBuilder::post_process(std::vector<int> &obj_indices,
                               std::vector<int> &emitter_prims) {
    // remap all the vertices, normals, UVs and object indices for SBVH. There
    // are two major step for this: (1) reordered vertices, normals, UVs, object
    // index and sphere_flags using an multi-threading approach (or SIMD). (2)
    // Deal with the emissive primitives (remove duplication)
    size_t original_size = vertices[0].size();
    remap_helper_func(flattened_idxs, vertices);
    remap_helper_func(flattened_idxs, normals);
    remap_helper_func(flattened_idxs, uvs);
    remap_helper_func<std::vector<int>, 1>(flattened_idxs, obj_indices);
    remap_helper_func<std::vector<bool>, 1>(flattened_idxs, sphere_flags);

    const size_t num_prims = flattened_idxs.size();
    std::vector<std::vector<int>> eprim_idxs(num_emitters);
    std::vector<bool> visited(original_size, false);
    for (int i = 0; i < num_prims; i++) {
        // skip duplicated emissive primitives, if the duplicated primitives are
        // not skipped over, the emissive primitive sampling will be biased so
        // the emissive indices should be unique
        int origin_prim_id = flattened_idxs[i];
        if (visited[origin_prim_id])
            continue;
        visited[origin_prim_id] = true;

        int obj_idx = obj_indices[i] & 0x000fffff;
        const auto &object = objects[obj_idx];
        if (object.is_emitter()) {
            int emitter_idx = object.emitter_id - 1;
            eprim_idxs[emitter_idx].push_back(i);
        }
    }

    std::vector<int> e_prim_offsets;
    e_prim_offsets.push_back(0);
    for (const auto &eprim_idx : eprim_idxs) {
        e_prim_offsets.push_back(eprim_idx.size());
        for (int index : eprim_idx)
            emitter_prims.push_back(index);
    }
    std::partial_sum(e_prim_offsets.begin(), e_prim_offsets.end(),
                     e_prim_offsets.begin());
    for (ObjInfo &obj : objects) {
        if (!obj.is_emitter())
            continue;
        obj.prim_offset = e_prim_offsets[obj.emitter_id - 1];
    }
}

// Try to use two threads to build the BVH
void SBVHBuilder::build(const std::vector<int> &obj_med_idxs,
                        const Vec3 &world_min, const Vec3 &world_max,
                        std::vector<int> &obj_idxs, std::vector<float4> &nodes,
                        std::vector<CompactNode> &cache_nodes,
                        int &cache_max_level) {
    const auto &points1 = vertices[0], &points2 = vertices[1],
               &points3 = vertices[2];

    std::vector<PrimMappingInfo> idx_prs;
    std::vector<BVHInfo> bvh_infos;
    int node_num = 0, num_prims_all = points1.size();
    BVHBuilder::index_input(objects, sphere_flags, idx_prs, num_prims_all);
    BVHBuilder::create_bvh_info(points1, points2, points3, idx_prs,
                                obj_med_idxs, bvh_infos);

    // spatial split almost always ends up with more primitives
    flattened_idxs.reserve(num_prims_all * 2);
    SBVHNode *root_node =
        sbvh_root_start(points1, points2, points3, world_min, world_max,
                        flattened_idxs, bvh_infos, node_num, max_prim_node);

    printf("[SBVH] SBVH tree max depth: %d, duplicated primitives: %d (%d)\n",
           max_depth, flattened_idxs.size(), points1.size());
    cache_max_level = std::min(std::max(max_depth - 1, 0), cache_max_level);
    nodes.reserve(node_num << 1);
    cache_nodes.reserve(1 << cache_max_level);

    recursive_linearize(root_node, nodes, cache_nodes, 0);
    printf("[SBVH] Number of nodes to cache: %lu (%d)\n", cache_nodes.size(),
           cache_max_level);

    obj_idxs.reserve(bvh_infos.size());
    for (BVHInfo &bvh : bvh_infos) {
        obj_idxs.emplace_back(bvh.bound.__bytes1);
    }
    delete root_node;
}
