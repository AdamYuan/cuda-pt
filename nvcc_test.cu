#include <iostream>
#include <hip/hip_runtime.h>

class Vec3 {
private:
    float3 _data;
public:
    __host__ __device__ Vec3() {}
    constexpr __host__ __device__
    Vec3(float _x, float _y, float _z): 
        _data({_x, _y, _z}) {}

    constexpr __host__ __device__ const float& x() const { return _data.x; }
    constexpr __host__ __device__ const float& y() const { return _data.y; }
    constexpr __host__ __device__ const float& z() const { return _data.z; }
};

int main() {
    hipDeviceProp_t prop;
    int device = 0;
    hipGetDeviceProperties(&prop, device);
    std::cout << "sharedMemPerBlock: " << prop.sharedMemPerBlock << " bytes" << std::endl;
    std::cout << "sharedMemPerMultiprocessor: " << prop.sharedMemPerMultiprocessor << " bytes" << std::endl;
    std::cout << "sharedMemPerBlockOptin: " << prop.sharedMemPerBlockOptin << " bytes" << std::endl;
    std::cout << "maxThreadsPerBlock: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "maxThreadsPerMultiProcessor: " << prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "concurrentKernels: " << prop.concurrentKernels << std::endl;
    std::cout << "maxBlocksPerMultiProcessor: " << prop.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "multiProcessorCount: " << prop.multiProcessorCount << std::endl;
    std::cout << "totalConstMem: " << prop.totalConstMem << " bytes"<< std::endl;

    constexpr Vec3 data(1, 2, 3);
    std::cout << data.x() << ", " << data.y() << ", " << data.z() << std::endl;
    return 0;
}